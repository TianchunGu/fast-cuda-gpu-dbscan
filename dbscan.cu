#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <time.h>

#include <algorithm>
#include <ctime>
#include <fstream>
#include <map>
#include <math.h>
#include <set>
#include <vector>

#include "common.h"
#include "indexing.h"

__device__ void searchPoints(double *data, int chainID, double *dataset,
                             int *results, struct IndexStructure **indexBuckets,
                             int *indexesStack, int *dataValue,
                             double *upperBounds, double *binWidth) {
  __shared__ int resultsCount;
  __shared__ int indexBucketSize;
  __shared__ int currentIndex;
  __shared__ int currentIndexSize;
  __shared__ double comparingData;

  if (threadIdx.x == 0) {
    resultsCount = 0;
    indexBucketSize = 1;
    for (int i = 0; i < DIMENSION; i++) {
      indexBucketSize *= 3;
    }
    indexBucketSize = indexBucketSize * chainID;
    currentIndexSize = indexBucketSize;
    indexesStack[currentIndexSize++] = 0;
  }
  __syncthreads();

  while (currentIndexSize > indexBucketSize) {
    if (threadIdx.x == 0) {
      currentIndexSize = currentIndexSize - 1;
      currentIndex = indexesStack[currentIndexSize];
      comparingData = data[indexBuckets[currentIndex]->dimension];
    }
    __syncthreads();

    for (int k = threadIdx.x + indexBuckets[currentIndex]->childFrom;
         k < indexBuckets[currentIndex]->childFrom + PARTITION_SIZE;
         k = k + THREAD_COUNT) {

      double leftRange;
      double rightRange;
      if (k == indexBuckets[currentIndex]->childFrom) {
        leftRange =
            upperBounds[k] - binWidth[indexBuckets[currentIndex]->dimension];
      } else {
        leftRange = upperBounds[k - 1];
      }

      rightRange = upperBounds[k];
      

      if (comparingData >= leftRange && comparingData < rightRange) {

        if (indexBuckets[currentIndex]->dimension == DIMENSION - 1) {
          int oldResultsCount = atomicAdd(&resultsCount, 1);
          results[chainID * POINTS_SEARCHED + oldResultsCount] = k;

          if (k > indexBuckets[currentIndex]->childFrom) {
            oldResultsCount = atomicAdd(&resultsCount, 1);
            results[chainID * POINTS_SEARCHED + oldResultsCount] = k - 1;
          }

          if (k < indexBuckets[currentIndex]->childFrom + PARTITION_SIZE - 1) {
            oldResultsCount = atomicAdd(&resultsCount, 1);
            results[chainID * POINTS_SEARCHED + oldResultsCount] = k + 1;
          }
        } else {
          int oldCurrentIndexSize = atomicAdd(&currentIndexSize, 1);
          indexesStack[oldCurrentIndexSize] = k;
          if (k > indexBuckets[currentIndex]->childFrom) {
            int oldCurrentIndexSize = atomicAdd(&currentIndexSize, 1);
            indexesStack[oldCurrentIndexSize] = k - 1;
          }
          if (k < indexBuckets[currentIndex]->childFrom + PARTITION_SIZE - 1) {
            int oldCurrentIndexSize = atomicAdd(&currentIndexSize, 1);
            indexesStack[oldCurrentIndexSize] = k + 1;
          }
        }
      }
    }

    __syncthreads();
  }
}


bool MonitorSeedPoints(vector<int> &unprocessedPoints,
                       map<int, set<int>> &collisionUnion, int *runningCluster,
                       int *d_cluster, int *d_seedList, int *d_seedLength,
                       int *d_collisionMatrix, int *d_extraCollision,
                       int *d_results) {
  /**
 **************************************************************************
 * Copy GPU variables content to CPU variables for seed list management
 **************************************************************************
 */
  int *localSeedLength;
  localSeedLength = (int *)malloc(sizeof(int) * THREAD_BLOCKS);
  gpuErrchk(hipMemcpy(localSeedLength, d_seedLength,
                       sizeof(int) * THREAD_BLOCKS, hipMemcpyDeviceToHost));

  int *localSeedList;
  localSeedList = (int *)malloc(sizeof(int) * THREAD_BLOCKS * MAX_SEEDS);
  gpuErrchk(hipMemcpy(localSeedList, d_seedList,
                       sizeof(int) * THREAD_BLOCKS * MAX_SEEDS,
                       hipMemcpyDeviceToHost));

  /**
 **************************************************************************
 * Copy GPU variables to CPU variables for collision detection
 **************************************************************************
 */

  int *localCollisionMatrix;
  localCollisionMatrix =
      (int *)malloc(sizeof(int) * THREAD_BLOCKS * THREAD_BLOCKS);
  gpuErrchk(hipMemcpy(localCollisionMatrix, d_collisionMatrix,
                       sizeof(int) * THREAD_BLOCKS * THREAD_BLOCKS,
                       hipMemcpyDeviceToHost));

  int *localExtraCollision;
  localExtraCollision =
      (int *)malloc(sizeof(int) * THREAD_BLOCKS * EXTRA_COLLISION_SIZE);
  gpuErrchk(hipMemcpy(localExtraCollision, d_extraCollision,
                       sizeof(int) * THREAD_BLOCKS * EXTRA_COLLISION_SIZE,
                       hipMemcpyDeviceToHost));

  /**
 **************************************************************************
 * If seedlist is empty and refill is also empty Then check the `
 * between chains and finalize the clusters
 **************************************************************************
 */

  int clusterMap[THREAD_BLOCKS];
  set<int> blockSet;
  for (int i = 0; i < THREAD_BLOCKS; i++) {
    blockSet.insert(i);
  }

  set<int>::iterator it;

  while (blockSet.empty() == 0) {
    it = blockSet.begin();
    int curBlock = *it;
    set<int> expansionQueue;
    set<int> finalQueue;

    expansionQueue.insert(curBlock);
    finalQueue.insert(curBlock);

    while (expansionQueue.empty() == 0) {
      it = expansionQueue.begin();
      int expandBlock = *it;
      expansionQueue.erase(it);
      blockSet.erase(expandBlock);
      for (int x = 0; x < THREAD_BLOCKS; x++) {
        if (x == expandBlock) continue;
        if (localCollisionMatrix[expandBlock * THREAD_BLOCKS + x] == 1 &&
            blockSet.find(x) != blockSet.end()) {
          expansionQueue.insert(x);
          finalQueue.insert(x);
        }
      }
    }

    for (it = finalQueue.begin(); it != finalQueue.end(); ++it) {
      clusterMap[*it] = curBlock;
    }
  }

  int clusterCountMap[THREAD_BLOCKS];
  for (int x = 0; x < THREAD_BLOCKS; x++) {
    clusterCountMap[x] = UNPROCESSED;
  }

  for (int x = 0; x < THREAD_BLOCKS; x++) {
    if (clusterCountMap[clusterMap[x]] != UNPROCESSED) continue;
    clusterCountMap[clusterMap[x]] = (*runningCluster);
    (*runningCluster)++;
  }

  for(int x = 0; x < THREAD_BLOCKS; x++) {
    thrust::replace(thrust::device, d_cluster, d_cluster + DATASET_COUNT, x, clusterCountMap[clusterMap[x]]);
  }

  for (int x = 0; x < THREAD_BLOCKS; x++) {
    if (localExtraCollision[x * EXTRA_COLLISION_SIZE] == -1) continue;
    int minCluster = localExtraCollision[x * EXTRA_COLLISION_SIZE];
    thrust::replace(thrust::device, d_cluster, d_cluster + DATASET_COUNT,
                    clusterCountMap[clusterMap[x]], minCluster);
    for (int y = 0; y < EXTRA_COLLISION_SIZE; y++) {
      if (localExtraCollision[x * EXTRA_COLLISION_SIZE + y] == UNPROCESSED)
        break;
      int data = localExtraCollision[x * EXTRA_COLLISION_SIZE + y];
      thrust::replace(thrust::device, d_cluster, d_cluster + DATASET_COUNT,
                      data, minCluster);
    }
  }

  /**
 **************************************************************************
 * After finilazing the cluster, check the remaining points and
 * insert one point to each of the seedlist
 **************************************************************************
 */

 int *localCluster;
 localCluster = (int *)malloc(sizeof(int) * DATASET_COUNT);
 gpuErrchk(hipMemcpy(localCluster, d_cluster, sizeof(int) * DATASET_COUNT,
                      hipMemcpyDeviceToHost));

  int complete = 0;
  for (int i = 0; i < THREAD_BLOCKS; i++) {
    bool found = false;
    while (!unprocessedPoints.empty()) {
      int lastPoint = unprocessedPoints.back();
      unprocessedPoints.pop_back();

      if (localCluster[lastPoint] == UNPROCESSED) {
        localSeedLength[i] = 1;
        localSeedList[i * MAX_SEEDS] = lastPoint;
        found = true;
        break;
      }
    }

    if (!found) {
      complete++;
    }
  }

  /**
**************************************************************************
* FInally, transfer back the CPU memory to GPU and run DBSCAN process
**************************************************************************
*/

  gpuErrchk(hipMemcpy(d_seedLength, localSeedLength,
                       sizeof(int) * THREAD_BLOCKS, hipMemcpyHostToDevice));

  gpuErrchk(hipMemcpy(d_seedList, localSeedList,
                       sizeof(int) * THREAD_BLOCKS * MAX_SEEDS,
                       hipMemcpyHostToDevice));

  /**
 **************************************************************************
 * Free CPU memory allocations
 **************************************************************************
 */

  free(localCluster);
  free(localSeedList);
  free(localSeedLength);
  free(localCollisionMatrix);
  free(localExtraCollision);

  if (complete == THREAD_BLOCKS) {
    return true;
  }

  return false;
}

/**
**************************************************************************
//////////////////////////////////////////////////////////////////////////
* Get DBSCAN result
* Get the final cluster and print the overall result
//////////////////////////////////////////////////////////////////////////
**************************************************************************
*/
void GetDbscanResult(int *d_cluster, int *runningCluster, int *clusterCount,
                     int *noiseCount) {
  int localClusterCount = 0;
  
  for (int i = THREAD_BLOCKS; i <= (*runningCluster); i++) {
    if (thrust::find(thrust::device, d_cluster, d_cluster + DATASET_COUNT, i) !=
        d_cluster + DATASET_COUNT) {
        localClusterCount++;
    }
  }
  *clusterCount = localClusterCount;
  *noiseCount = thrust::count(thrust::device, d_cluster,
                              d_cluster + DATASET_COUNT, NOISE);
}

/**
**************************************************************************
//////////////////////////////////////////////////////////////////////////
* Mark as candidate
* It does the following functions:
* 1) Mark the neighbor's cluster with chainID if its old state is unprocessed
* 2) If the oldstate is unprocessed, insert the neighnor point to seed list
* 3) if the seed list exceeds max value, insert into refill seed list
* 4) If the old state is less than THREAD BLOCK, record the collision in
* collision matrix
* 5) If the old state is greater than THREAD BLOCK, record the collision
* in extra collision
//////////////////////////////////////////////////////////////////////////
**************************************************************************
*/

__device__ void MarkAsCandidate(int neighborID, int chainID, int *cluster,
                                int *seedList, int *seedLength,
                                int *collisionMatrix, int *extraCollision) {
  /**
**************************************************************************
* Get the old cluster state of the neighbor
* If the state is unprocessed, assign it with chainID
**************************************************************************
*/
  register int oldState =
      atomicCAS(&(cluster[neighborID]), UNPROCESSED, chainID);

  /**
 **************************************************************************
 * For unprocessed old state of neighbors, add them to seedlist and
 * refill seedlist
 **************************************************************************
 */
  if (oldState == UNPROCESSED) {
    register int sl = atomicAdd(&(seedLength[chainID]), 1);
    if (sl < MAX_SEEDS) {
      seedList[chainID * MAX_SEEDS + sl] = neighborID;
    }
  }

  /**
 **************************************************************************
 * If the old state is greater than thread block, record the extra collisions
 **************************************************************************
 */

  else if (oldState >= THREAD_BLOCKS) {
    for (int i = 0; i < EXTRA_COLLISION_SIZE; i++) {
      register int changedState =
          atomicCAS(&(extraCollision[chainID * EXTRA_COLLISION_SIZE + i]),
                    UNPROCESSED, oldState);
      if (changedState == UNPROCESSED || changedState == oldState) {
        break;
      }
    }
  }

  /**
 **************************************************************************
 * If the old state of neighbor is not noise, not member of chain and cluster
 * is within THREADBLOCK, maek the collision between old and new state
 **************************************************************************
 */
  else if (oldState != NOISE && oldState != chainID &&
           oldState < THREAD_BLOCKS) {
    collisionMatrix[oldState * THREAD_BLOCKS + chainID] = 1;
    collisionMatrix[chainID * THREAD_BLOCKS + oldState] = 1;
  }

  /**
 **************************************************************************
 * If the old state is noise, assign it to chainID cluster
 **************************************************************************
 */
  else if (oldState == NOISE) {
    oldState = atomicCAS(&(cluster[neighborID]), NOISE, chainID);
  }
}


/**
**************************************************************************
//////////////////////////////////////////////////////////////////////////
* DBSCAN: Main kernel function of the algorithm
* It does the following functions.
* 1) Every block gets a point from seedlist to expand. If these points are
* processed already, it returns
* 2) It expands the points by finding neighbors points
* 3) Checks for the collision and mark the collision in collision matrix
//////////////////////////////////////////////////////////////////////////
**************************************************************************
*/
__global__ void DBSCAN(double *dataset, int *cluster, int *seedList,
                       int *seedLength, int *collisionMatrix,
                       int *extraCollision, int *results,
                       struct IndexStructure **indexBuckets,

                       int *indexesStack, int *dataValue, double *upperBounds,
                       double *binWidth) {
  /**
 **************************************************************************
 * Define shared variables
 **************************************************************************
 */

  // Point ID to expand by a block
  __shared__ int pointID;

  // Neighbors to store of neighbors points exceeds minpoints
  __shared__ int neighborBuffer[MINPTS];

  // It counts the total neighbors
  __shared__ int neighborCount;

  // ChainID is basically blockID
  __shared__ int chainID;

  // Store the point from pointID
  __shared__ double point[DIMENSION];

  // Length of the seedlist to check its size
  __shared__ int currentSeedLength;

  __shared__ int resultId;

  if (threadIdx.x == 0) {
    chainID = blockIdx.x;
    currentSeedLength = seedLength[chainID];
    pointID = seedList[chainID * MAX_SEEDS + currentSeedLength - 1];
  }
  __syncthreads();

  int threadId = blockDim.x * blockIdx.x + threadIdx.x;
  for (int x = threadId; x < THREAD_BLOCKS * THREAD_BLOCKS;
       x = x + THREAD_BLOCKS * THREAD_COUNT) {
    collisionMatrix[x] = UNPROCESSED;
  }
  for (int x = threadId; x < THREAD_BLOCKS * EXTRA_COLLISION_SIZE;
       x = x + THREAD_BLOCKS * THREAD_COUNT) {
    extraCollision[x] = UNPROCESSED;
  }

  __syncthreads();

  // Complete the seedlist to proceed.

  while (seedLength[chainID] != 0) {
    for (int x = threadId; x < THREAD_BLOCKS * POINTS_SEARCHED;
         x = x + THREAD_BLOCKS * THREAD_COUNT) {
      results[x] = UNPROCESSED;
    }
    __syncthreads();

    // Assign chainID, current seed length and pointID
    if (threadIdx.x == 0) {
      chainID = blockIdx.x;
      currentSeedLength = seedLength[chainID];
      pointID = seedList[chainID * MAX_SEEDS + currentSeedLength - 1];
    }
    __syncthreads();

    // Check if the point is already processed
    if (threadIdx.x == 0) {
      seedLength[chainID] = currentSeedLength - 1;
      neighborCount = 0;
      for (int x = 0; x < DIMENSION; x++) {
        point[x] = dataset[pointID * DIMENSION + x];
      }
    }
    __syncthreads();

    /**
**************************************************************************
* Find the neighbors of the pointID
* Mark point as candidate if points are more than min points
* Keep record of left over neighbors in neighborBuffer
**************************************************************************
*/

    searchPoints(point, chainID, dataset, results, indexBuckets, indexesStack,
                 dataValue, upperBounds, binWidth);

    __syncthreads();

    for (int k = 0; k < POINTS_SEARCHED; k++) {
      if (threadIdx.x == 0) {
        resultId = results[chainID * POINTS_SEARCHED + k];
      }
      __syncthreads();

      if (resultId == -1) break;

      for (int i = threadIdx.x + indexBuckets[resultId]->dataBegin;
           i < indexBuckets[resultId]->dataEnd; i = i + THREAD_COUNT) {
        register double comparingPoint[DIMENSION];

        for (int x = 0; x < DIMENSION; x++) {
          comparingPoint[x] = dataset[dataValue[i] * DIMENSION + x];
        }

        register double distance = 0;
        for (int x = 0; x < DIMENSION; x++) {
          distance +=
              (point[x] - comparingPoint[x]) * (point[x] - comparingPoint[x]);
        }

        if (distance <= EPS * EPS) {
          register int currentNeighborCount = atomicAdd(&neighborCount, 1);
          if (currentNeighborCount >= MINPTS) {
            MarkAsCandidate(dataValue[i], chainID, cluster, seedList,
                            seedLength, collisionMatrix, extraCollision);
          } else {
            neighborBuffer[currentNeighborCount] = dataValue[i];
          }
        }
      }
      __syncthreads();
    }
    __syncthreads();

    /**
**************************************************************************
* Mark the left over neighbors in neighborBuffer as cluster member
* If neighbors are less than MINPTS, assign pointID with noise
**************************************************************************
*/

    if (neighborCount >= MINPTS) {
      cluster[pointID] = chainID;
      for (int i = threadIdx.x; i < MINPTS; i = i + THREAD_COUNT) {
        MarkAsCandidate(neighborBuffer[i], chainID, cluster, seedList,
                        seedLength, collisionMatrix, extraCollision);
      }
    } else {
      cluster[pointID] = NOISE;
    }

    __syncthreads();

    /**
**************************************************************************
* Check Thread length, If it exceeds MAX limit the length
* As seedlist wont have data beyond its max length
**************************************************************************
*/

    if (threadIdx.x == 0 && seedLength[chainID] >= MAX_SEEDS) {
      seedLength[chainID] = MAX_SEEDS - 1;
    }
    __syncthreads();
  }
}