#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <time.h>

#include <algorithm>
#include <ctime>
#include <fstream>
#include <map>
#include <math.h>
#include <set>
#include <vector>

#include "common.h"
#include "indexing.h"
#include "dbscan.h"

__global__ void DBSCAN(double *dataset, int *cluster, int *seedList,
                       int *seedLength, int *collisionMatrix,
                       int *extraCollision, int *results,
                       struct IndexStructure **indexBuckets,

                       int *indexesStack, int *dataValue, double *upperBounds,
                       double *binWidth) {
  // Point ID to expand by a block
  __shared__ int pointID;

  // Neighbors to store of neighbors points exceeds minpoints
  __shared__ int neighborBuffer[MINPTS];

  // It counts the total neighbors
  __shared__ int neighborCount;

  // ChainID is basically blockID
  __shared__ int chainID;

  // Store the point from pointID
  __shared__ double point[DIMENSION];

  // Length of the seedlist to check its size
  __shared__ int currentSeedLength;

  __shared__ int resultId;

  if (threadIdx.x == 0) {
    chainID = blockIdx.x;
    currentSeedLength = seedLength[chainID];
    pointID = seedList[chainID * MAX_SEEDS + currentSeedLength - 1];
  }
  __syncthreads();

  int threadId = blockDim.x * blockIdx.x + threadIdx.x;
  for (int x = threadId; x < THREAD_BLOCKS * THREAD_BLOCKS;
       x = x + THREAD_BLOCKS * THREAD_COUNT) {
    collisionMatrix[x] = UNPROCESSED;
  }
  for (int x = threadId; x < THREAD_BLOCKS * EXTRA_COLLISION_SIZE;
       x = x + THREAD_BLOCKS * THREAD_COUNT) {
    extraCollision[x] = UNPROCESSED;
  }

  __syncthreads();

  // Complete the seedlist to proceed.

  while (seedLength[chainID] != 0) {
    for (int x = threadId; x < THREAD_BLOCKS * POINTS_SEARCHED;
         x = x + THREAD_BLOCKS * THREAD_COUNT) {
      results[x] = UNPROCESSED;
    }
    __syncthreads();

    // Assign chainID, current seed length and pointID
    if (threadIdx.x == 0) {
      chainID = blockIdx.x;
      currentSeedLength = seedLength[chainID];
      pointID = seedList[chainID * MAX_SEEDS + currentSeedLength - 1];
    }
    __syncthreads();

    // Check if the point is already processed
    if (threadIdx.x == 0) {
      seedLength[chainID] = currentSeedLength - 1;
      neighborCount = 0;
      for (int x = 0; x < DIMENSION; x++) {
        point[x] = dataset[pointID * DIMENSION + x];
      }
    }
    __syncthreads();

    ///////////////////////////////////////////////////////////////////////////////////

    searchPoints(point, chainID, dataset, results, indexBuckets, indexesStack,
                 dataValue, upperBounds, binWidth);

    __syncthreads();

    for (int k = 0; k < POINTS_SEARCHED; k++) {
      if (threadIdx.x == 0) {
        resultId = results[chainID * POINTS_SEARCHED + k];
      }
      __syncthreads();

      if (resultId == -1) break;

      for (int i = threadIdx.x + indexBuckets[resultId]->dataBegin;
           i < indexBuckets[resultId]->dataEnd; i = i + THREAD_COUNT) {
        register double comparingPoint[DIMENSION];

        for (int x = 0; x < DIMENSION; x++) {
          comparingPoint[x] = dataset[dataValue[i] * DIMENSION + x];
        }

        register double distance = 0;
        for (int x = 0; x < DIMENSION; x++) {
          distance +=
              (point[x] - comparingPoint[x]) * (point[x] - comparingPoint[x]);
        }

        if (distance <= EPS * EPS) {
          register int currentNeighborCount = atomicAdd(&neighborCount, 1);
          if (currentNeighborCount >= MINPTS) {
            MarkAsCandidate(dataValue[i], chainID, cluster, seedList,
                            seedLength, collisionMatrix, extraCollision);
          } else {
            neighborBuffer[currentNeighborCount] = dataValue[i];
          }
        }
      }
      __syncthreads();
    }
    __syncthreads();

    ///////////////////////////////////////////////////////////////////////////////////

    if (neighborCount >= MINPTS) {
      cluster[pointID] = chainID;
      for (int i = threadIdx.x; i < MINPTS; i = i + THREAD_COUNT) {
        MarkAsCandidate(neighborBuffer[i], chainID, cluster, seedList,
                        seedLength, collisionMatrix, extraCollision);
      }
    } else {
      cluster[pointID] = NOISE;
    }

    __syncthreads();
    ///////////////////////////////////////////////////////////////////////////////////

    if (threadIdx.x == 0 && seedLength[chainID] >= MAX_SEEDS) {
      seedLength[chainID] = MAX_SEEDS - 1;
    }
    __syncthreads();
  }
}

bool MonitorSeedPoints(vector<int> &unprocessedPoints, int *runningCluster,
                       int *d_cluster, int *d_seedList, int *d_seedLength,
                       int *d_collisionMatrix, int *d_extraCollision,
                       int *d_results) {
  int *localSeedLength;
  localSeedLength = (int *)malloc(sizeof(int) * THREAD_BLOCKS);
  gpuErrchk(hipMemcpy(localSeedLength, d_seedLength,
                       sizeof(int) * THREAD_BLOCKS, hipMemcpyDeviceToHost));

  int *localSeedList;
  localSeedList = (int *)malloc(sizeof(int) * THREAD_BLOCKS * MAX_SEEDS);
  gpuErrchk(hipMemcpy(localSeedList, d_seedList,
                       sizeof(int) * THREAD_BLOCKS * MAX_SEEDS,
                       hipMemcpyDeviceToHost));

  int *localCollisionMatrix;
  localCollisionMatrix =
      (int *)malloc(sizeof(int) * THREAD_BLOCKS * THREAD_BLOCKS);
  gpuErrchk(hipMemcpy(localCollisionMatrix, d_collisionMatrix,
                       sizeof(int) * THREAD_BLOCKS * THREAD_BLOCKS,
                       hipMemcpyDeviceToHost));

  int *localExtraCollision;
  localExtraCollision =
      (int *)malloc(sizeof(int) * THREAD_BLOCKS * EXTRA_COLLISION_SIZE);
  gpuErrchk(hipMemcpy(localExtraCollision, d_extraCollision,
                       sizeof(int) * THREAD_BLOCKS * EXTRA_COLLISION_SIZE,
                       hipMemcpyDeviceToHost));

  ////////////////////////////////////////////////////////////////////////////////////////

  int clusterMap[THREAD_BLOCKS];
  set<int> blockSet;
  for (int i = 0; i < THREAD_BLOCKS; i++) {
    blockSet.insert(i);
  }

  set<int>::iterator it;

  while (blockSet.empty() == 0) {
    it = blockSet.begin();
    int curBlock = *it;
    set<int> expansionQueue;
    set<int> finalQueue;

    expansionQueue.insert(curBlock);
    finalQueue.insert(curBlock);

    while (expansionQueue.empty() == 0) {
      it = expansionQueue.begin();
      int expandBlock = *it;
      expansionQueue.erase(it);
      blockSet.erase(expandBlock);
      for (int x = 0; x < THREAD_BLOCKS; x++) {
        if (x == expandBlock) continue;
        if (localCollisionMatrix[expandBlock * THREAD_BLOCKS + x] == 1 &&
            blockSet.find(x) != blockSet.end()) {
          expansionQueue.insert(x);
          finalQueue.insert(x);
        }
      }
    }

    for (it = finalQueue.begin(); it != finalQueue.end(); ++it) {
      clusterMap[*it] = curBlock;
    }
  }

  int clusterCountMap[THREAD_BLOCKS];
  for (int x = 0; x < THREAD_BLOCKS; x++) {
    clusterCountMap[x] = UNPROCESSED;
  }

  for (int x = 0; x < THREAD_BLOCKS; x++) {
    if (clusterCountMap[clusterMap[x]] != UNPROCESSED) continue;
    clusterCountMap[clusterMap[x]] = (*runningCluster);
    (*runningCluster)++;
  }

  for (int x = 0; x < THREAD_BLOCKS; x++) {
    thrust::replace(thrust::device, d_cluster, d_cluster + DATASET_COUNT, x,
                    clusterCountMap[clusterMap[x]]);
  }

  for (int x = 0; x < THREAD_BLOCKS; x++) {
    if (localExtraCollision[x * EXTRA_COLLISION_SIZE] == -1) continue;
    int minCluster = localExtraCollision[x * EXTRA_COLLISION_SIZE];
    thrust::replace(thrust::device, d_cluster, d_cluster + DATASET_COUNT,
                    clusterCountMap[clusterMap[x]], minCluster);
    for (int y = 0; y < EXTRA_COLLISION_SIZE; y++) {
      if (localExtraCollision[x * EXTRA_COLLISION_SIZE + y] == UNPROCESSED)
        break;
      int data = localExtraCollision[x * EXTRA_COLLISION_SIZE + y];
      thrust::replace(thrust::device, d_cluster, d_cluster + DATASET_COUNT,
                      data, minCluster);
    }
  }

  //////////////////////////////////////////////////////////////////////////////////////////

  int *localCluster;
  localCluster = (int *)malloc(sizeof(int) * DATASET_COUNT);
  gpuErrchk(hipMemcpy(localCluster, d_cluster, sizeof(int) * DATASET_COUNT,
                       hipMemcpyDeviceToHost));

  int complete = 0;
  for (int i = 0; i < THREAD_BLOCKS; i++) {
    bool found = false;
    while (!unprocessedPoints.empty()) {
      int lastPoint = unprocessedPoints.back();
      unprocessedPoints.pop_back();

      if (localCluster[lastPoint] == UNPROCESSED) {
        localSeedLength[i] = 1;
        localSeedList[i * MAX_SEEDS] = lastPoint;
        found = true;
        break;
      }
    }

    if (!found) {
      complete++;
    }
  }

  // FInally, transfer back the CPU memory to GPU and run DBSCAN process

  gpuErrchk(hipMemcpy(d_seedLength, localSeedLength,
                       sizeof(int) * THREAD_BLOCKS, hipMemcpyHostToDevice));

  gpuErrchk(hipMemcpy(d_seedList, localSeedList,
                       sizeof(int) * THREAD_BLOCKS * MAX_SEEDS,
                       hipMemcpyHostToDevice));

  // Free CPU memories

  free(localCluster);
  free(localSeedList);
  free(localSeedLength);
  free(localCollisionMatrix);
  free(localExtraCollision);

  if (complete == THREAD_BLOCKS) {
    return true;
  }

  return false;
}

__device__ void MarkAsCandidate(int neighborID, int chainID, int *cluster,
                                int *seedList, int *seedLength,
                                int *collisionMatrix, int *extraCollision) {
  register int oldState =
      atomicCAS(&(cluster[neighborID]), UNPROCESSED, chainID);

  if (oldState == UNPROCESSED) {
    register int sl = atomicAdd(&(seedLength[chainID]), 1);
    if (sl < MAX_SEEDS) {
      seedList[chainID * MAX_SEEDS + sl] = neighborID;
    }
  }

  else if (oldState >= THREAD_BLOCKS) {
    for (int i = 0; i < EXTRA_COLLISION_SIZE; i++) {
      register int changedState =
          atomicCAS(&(extraCollision[chainID * EXTRA_COLLISION_SIZE + i]),
                    UNPROCESSED, oldState);
      if (changedState == UNPROCESSED || changedState == oldState) {
        break;
      }
    }
  }

  else if (oldState != NOISE && oldState != chainID &&
           oldState < THREAD_BLOCKS) {
    collisionMatrix[oldState * THREAD_BLOCKS + chainID] = 1;
    collisionMatrix[chainID * THREAD_BLOCKS + oldState] = 1;
  }

  else if (oldState == NOISE) {
    oldState = atomicCAS(&(cluster[neighborID]), NOISE, chainID);
  }
}

__device__ void searchPoints(double *data, int chainID, double *dataset,
                             int *results, struct IndexStructure **indexBuckets,
                             int *indexesStack, int *dataValue,
                             double *upperBounds, double *binWidth) {
  __shared__ int resultsCount;
  __shared__ int indexBucketSize;
  __shared__ int currentIndex;
  __shared__ int currentIndexSize;
  __shared__ double comparingData;

  if (threadIdx.x == 0) {
    resultsCount = 0;
    indexBucketSize = 1;
    for (int i = 0; i < DIMENSION; i++) {
      indexBucketSize *= 3;
    }
    indexBucketSize = indexBucketSize * chainID;
    currentIndexSize = indexBucketSize;
    indexesStack[currentIndexSize++] = 0;
  }
  __syncthreads();

  while (currentIndexSize > indexBucketSize) {
    if (threadIdx.x == 0) {
      currentIndexSize = currentIndexSize - 1;
      currentIndex = indexesStack[currentIndexSize];
      comparingData = data[indexBuckets[currentIndex]->dimension];
    }
    __syncthreads();

    for (int k = threadIdx.x + indexBuckets[currentIndex]->childFrom;
         k < indexBuckets[currentIndex]->childFrom + PARTITION_SIZE;
         k = k + THREAD_COUNT) {
      double leftRange;
      double rightRange;
      if (k == indexBuckets[currentIndex]->childFrom) {
        leftRange =
            upperBounds[k] - binWidth[indexBuckets[currentIndex]->dimension];
      } else {
        leftRange = upperBounds[k - 1];
      }

      rightRange = upperBounds[k];

      if (comparingData >= leftRange && comparingData < rightRange) {
        if (indexBuckets[currentIndex]->dimension == DIMENSION - 1) {
          int oldResultsCount = atomicAdd(&resultsCount, 1);
          results[chainID * POINTS_SEARCHED + oldResultsCount] = k;

          if (k > indexBuckets[currentIndex]->childFrom) {
            oldResultsCount = atomicAdd(&resultsCount, 1);
            results[chainID * POINTS_SEARCHED + oldResultsCount] = k - 1;
          }

          if (k < indexBuckets[currentIndex]->childFrom + PARTITION_SIZE - 1) {
            oldResultsCount = atomicAdd(&resultsCount, 1);
            results[chainID * POINTS_SEARCHED + oldResultsCount] = k + 1;
          }
        } else {
          int oldCurrentIndexSize = atomicAdd(&currentIndexSize, 1);
          indexesStack[oldCurrentIndexSize] = k;
          if (k > indexBuckets[currentIndex]->childFrom) {
            int oldCurrentIndexSize = atomicAdd(&currentIndexSize, 1);
            indexesStack[oldCurrentIndexSize] = k - 1;
          }
          if (k < indexBuckets[currentIndex]->childFrom + PARTITION_SIZE - 1) {
            int oldCurrentIndexSize = atomicAdd(&currentIndexSize, 1);
            indexesStack[oldCurrentIndexSize] = k + 1;
          }
        }
      }
    }

    __syncthreads();
  }
}

void GetDbscanResult(int *d_cluster, int *runningCluster, int *clusterCount,
                     int *noiseCount) {
  *noiseCount = thrust::count(thrust::device, d_cluster, d_cluster + DATASET_COUNT, NOISE);
  int *d_localCluster;
  gpuErrchk(hipMalloc((void **)&d_localCluster, sizeof(int) * DATASET_COUNT));
  thrust::copy(thrust::device, d_cluster, d_cluster + DATASET_COUNT, d_localCluster);
  thrust::sort(thrust::device, d_localCluster, d_localCluster + DATASET_COUNT);
  *clusterCount = thrust::unique(thrust::device, d_localCluster, d_localCluster + DATASET_COUNT) - d_localCluster - 1;
  hipFree(d_localCluster);
}

void TestGetDbscanResult(int *d_cluster, int *runningCluster, int *clusterCount,
                         int *noiseCount) {
  *noiseCount = thrust::count(thrust::device, d_cluster, d_cluster + DATASET_COUNT, NOISE);
  int *d_localCluster;
  gpuErrchk(hipMalloc((void **)&d_localCluster, sizeof(int) * DATASET_COUNT));
  thrust::copy(thrust::device, d_cluster, d_cluster + DATASET_COUNT, d_localCluster);
  thrust::sort(thrust::device, d_localCluster, d_localCluster + DATASET_COUNT);
  *clusterCount = thrust::unique(thrust::device, d_localCluster, d_localCluster + DATASET_COUNT) - d_localCluster - 1;
  hipFree(d_localCluster);

}

__global__ void DBSCAN_ONE_INSTANCE(double *dataset, int *cluster,
                                    int *seedList, int *seedLength,
                                    int *collisionMatrix, int *extraCollision,
                                    int *results,
                                    struct IndexStructure **indexBuckets,
                                    int *indexesStack, int *dataValue,
                                    double *upperBounds, double *binWidth) {
  // Point ID to expand by a block
  __shared__ int pointID;

  // Neighbors to store of neighbors points exceeds minpoints
  __shared__ int neighborBuffer[MINPTS];

  // It counts the total neighbors
  __shared__ int neighborCount;

  // ChainID is basically blockID
  __shared__ int chainID;

  // Store the point from pointID
  __shared__ double point[DIMENSION];

  // Length of the seedlist to check its size
  __shared__ int currentSeedLength;

  __shared__ int resultId;


  int threadId = blockDim.x * blockIdx.x + threadIdx.x;
  for (int x = threadId; x < THREAD_BLOCKS * THREAD_BLOCKS;
       x = x + THREAD_BLOCKS * THREAD_COUNT) {
    collisionMatrix[x] = UNPROCESSED;
  }
  for (int x = threadId; x < THREAD_BLOCKS * EXTRA_COLLISION_SIZE;
       x = x + THREAD_BLOCKS * THREAD_COUNT) {
    extraCollision[x] = UNPROCESSED;
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    chainID = blockIdx.x;
  }
  __syncthreads();

  while (seedLength[chainID] != 0) {
    for (int x = threadId; x < THREAD_BLOCKS * POINTS_SEARCHED;
         x = x + THREAD_BLOCKS * THREAD_COUNT) {
      results[x] = UNPROCESSED;
    }
    __syncthreads();

    // Assign chainID, current seed length and pointID
    if (threadIdx.x == 0) {
      currentSeedLength = seedLength[chainID];
      pointID = seedList[chainID * MAX_SEEDS + currentSeedLength - 1];
    }
    __syncthreads();

    // Check if the point is already processed
    if (threadIdx.x == 0) {
      seedLength[chainID] = currentSeedLength - 1;
      neighborCount = 0;
      for (int x = 0; x < DIMENSION; x++) {
        point[x] = dataset[pointID * DIMENSION + x];
      }
    }
    __syncthreads();

    ///////////////////////////////////////////////////////////////////////////////////

    searchPoints(point, chainID, dataset, results, indexBuckets, indexesStack,
                 dataValue, upperBounds, binWidth);

    __syncthreads();

    for (int k = 0; k < POINTS_SEARCHED; k++) {
      if (threadIdx.x == 0) {
        resultId = results[chainID * POINTS_SEARCHED + k];
      }
      __syncthreads();

      if (resultId == -1) break;

      for (int i = threadIdx.x + indexBuckets[resultId]->dataBegin;
           i < indexBuckets[resultId]->dataEnd; i = i + THREAD_COUNT) {
        register double comparingPoint[DIMENSION];

        for (int x = 0; x < DIMENSION; x++) {
          comparingPoint[x] = dataset[dataValue[i] * DIMENSION + x];
        }

        register double distance = 0;
        for (int x = 0; x < DIMENSION; x++) {
          distance +=
              (point[x] - comparingPoint[x]) * (point[x] - comparingPoint[x]);
        }

        if (distance <= EPS * EPS) {
          register int currentNeighborCount = atomicAdd(&neighborCount, 1);
          if (currentNeighborCount >= MINPTS) {
            MarkAsCandidate(dataValue[i], chainID, cluster, seedList,
                            seedLength, collisionMatrix, extraCollision);
          } else {
            neighborBuffer[currentNeighborCount] = dataValue[i];
          }
        }
      }
      __syncthreads();
    }
    __syncthreads();

    ///////////////////////////////////////////////////////////////////////////////////

    if (neighborCount >= MINPTS) {
      cluster[pointID] = chainID;
      for (int i = threadIdx.x; i < MINPTS; i = i + THREAD_COUNT) {
        MarkAsCandidate(neighborBuffer[i], chainID, cluster, seedList,
                        seedLength, collisionMatrix, extraCollision);
      }
    } else {
      cluster[pointID] = NOISE;
    }

    __syncthreads();
    ///////////////////////////////////////////////////////////////////////////////////

    if (threadIdx.x == 0 && seedLength[chainID] >= MAX_SEEDS) {
      seedLength[chainID] = MAX_SEEDS - 1;
    }
    __syncthreads();
  }
}



__global__ void COLLISION_DETECTION(int *collisionMatrix, int *extraCollision,
                                    int *cluster, int *clusterMap, int*clusterCountMap, int* runningCluster) {
  if (threadIdx.x == 0) {
    clusterMap[blockIdx.x] = blockIdx.x;
    clusterCountMap[blockIdx.x] = UNPROCESSED;
  }
  __syncthreads();

  if (blockIdx.x == 0) {

    __shared__ int blockSet[THREAD_BLOCKS];
    __shared__ int blocksetCount;
    __shared__ int curBlock;
    __shared__ int expansionQueue[THREAD_BLOCKS];
    __shared__ int finalQueue[THREAD_BLOCKS];
    __shared__ int expansionQueueCount;
    __shared__ int finalQueueCount;
    __shared__ int expandBlock;
    

    for (int i = threadIdx.x; i < THREAD_BLOCKS; i = i + THREAD_COUNT) {
      blockSet[i] = i;
    }
    __syncthreads();

    if (threadIdx.x == 0) {
      blocksetCount = THREAD_BLOCKS;
    }
    __syncthreads();

    while (blocksetCount > 0) {
      if (threadIdx.x == 0) {
        curBlock = blockSet[0];
        expansionQueueCount = 0;
        finalQueueCount = 0;
        expansionQueue[expansionQueueCount++] = curBlock;
        finalQueue[finalQueueCount++] = curBlock;
      }
      __syncthreads();

      while (expansionQueueCount > 0) {
        if (threadIdx.x == 0) {
          expandBlock = expansionQueue[--expansionQueueCount];

          thrust::remove(thrust::device, expansionQueue,
                         expansionQueue + THREAD_BLOCKS, expandBlock);
          thrust::remove(thrust::device, blockSet, blockSet + THREAD_BLOCKS,
                         expandBlock);
          blocksetCount--;
        }
        __syncthreads();

        for (int x = threadIdx.x; x < THREAD_BLOCKS; x = x + THREAD_COUNT) {
          if (x != expandBlock) {
            if (collisionMatrix[expandBlock * THREAD_BLOCKS + x] == 1 &&
                thrust::find(thrust::device, blockSet, blockSet + blocksetCount,
                             x) != blockSet + blocksetCount) {
              if (thrust::find(thrust::device, expansionQueue,
                               expansionQueue + expansionQueueCount,
                               x) == expansionQueue + expansionQueueCount) {
                int oldExpansionQueueCount = atomicAdd(&expansionQueueCount, 1);
                expansionQueue[oldExpansionQueueCount] = x;
              }

              if (thrust::find(thrust::device, finalQueue,
                               finalQueue + finalQueueCount,
                               x) == finalQueue + finalQueueCount) {
                int oldFinalQueueCount = atomicAdd(&finalQueueCount, 1);
                finalQueue[oldFinalQueueCount] = x;
              }
            }
          }

          __syncthreads();
        }
      };
      __syncthreads();

      for (int c = threadIdx.x; c < finalQueueCount; c = c + THREAD_COUNT) {
        clusterMap[finalQueue[c]] = curBlock;
      }
      __syncthreads();
    };

    if(threadIdx.x == 0) {
      for (int x = 0; x < THREAD_BLOCKS; x++) {
        if (clusterCountMap[clusterMap[x]] == UNPROCESSED) {
          clusterCountMap[clusterMap[x]] = runningCluster[0]++;
        }
      }
    }
    __syncthreads();

  }
  __syncthreads();
  
}

__global__ void COLLISION_MERGE(int *collisionMatrix, int *extraCollision,
  int *cluster,  int *clusterMap, int*clusterCountMap) {

  __shared__ int chainID;

  if (threadIdx.x == 0) {
    chainID = blockIdx.x;
  }
  __syncthreads();
  

  int threadId = blockDim.x * blockIdx.x + threadIdx.x;
  for (int i = threadId; i < DATASET_COUNT; i = i + THREAD_BLOCKS * THREAD_COUNT) {
    if (cluster[i] >= 0 && cluster[i] < THREAD_BLOCKS) {
      cluster[i] = clusterCountMap[clusterMap[cluster[i]]];
    }
  }
  __syncthreads();

  for (int y = 0; y < EXTRA_COLLISION_SIZE; y++) {
    if (extraCollision[chainID * EXTRA_COLLISION_SIZE + y] != UNPROCESSED) {
      for (int i = threadIdx.x; i < DATASET_COUNT; i = i + THREAD_COUNT) {
        if (extraCollision[chainID * EXTRA_COLLISION_SIZE + y] == cluster[i])
          cluster[i] = extraCollision[chainID * EXTRA_COLLISION_SIZE];
        if (y == 0 && cluster[i] == clusterCountMap[clusterMap[chainID]])
          cluster[i] = extraCollision[chainID * EXTRA_COLLISION_SIZE];
      }
    }
  }
  __syncthreads();
}


bool TestMonitorSeedPoints(vector<int> &unprocessedPoints,
                       int *d_cluster, int *d_seedList, int *d_seedLength,
                       int *d_collisionMatrix, int *d_extraCollision,
                       int *d_results, int *d_clusterMap, int *d_clusterCountMap, int* d_runningCluster) {
  int *localSeedLength;
  localSeedLength = (int *)malloc(sizeof(int) * THREAD_BLOCKS);
  gpuErrchk(hipMemcpy(localSeedLength, d_seedLength,
                       sizeof(int) * THREAD_BLOCKS, hipMemcpyDeviceToHost));

  int *localSeedList;
  localSeedList = (int *)malloc(sizeof(int) * THREAD_BLOCKS * MAX_SEEDS);
  gpuErrchk(hipMemcpy(localSeedList, d_seedList,
                       sizeof(int) * THREAD_BLOCKS * MAX_SEEDS,
                       hipMemcpyDeviceToHost));


  int completeSeedListFirst = false;
  for (int i = 0; i < THREAD_BLOCKS; i++) {
    if (localSeedLength[i] > 0) {
      completeSeedListFirst = true;
    }
  }
  if (completeSeedListFirst) {
    free(localSeedList);
    free(localSeedLength);
    return false;
  }

  //////////////////////////////////////////////////////////////////////////////////////////

  gpuErrchk(hipDeviceSynchronize());
  COLLISION_DETECTION<<<dim3(THREAD_BLOCKS, 1), dim3(THREAD_COUNT, 1)>>>(d_collisionMatrix, d_extraCollision,
    d_cluster, d_clusterMap, d_clusterCountMap, d_runningCluster);
  gpuErrchk(hipDeviceSynchronize());

  COLLISION_MERGE<<<dim3(THREAD_BLOCKS, 1), dim3(THREAD_COUNT, 1)>>>(d_collisionMatrix, d_extraCollision,
    d_cluster, d_clusterMap, d_clusterCountMap);
  gpuErrchk(hipDeviceSynchronize());

  //////////////////////////////////////////////////////////////////////////////////////////
  int* d_datasetSequence;
  gpuErrchk(hipMalloc((void **)&d_datasetSequence, sizeof(int) * DATASET_COUNT));
  int *d_tempCluster;
  gpuErrchk(hipMalloc((void **)&d_tempCluster, sizeof(int) * DATASET_COUNT));

  thrust::sequence(thrust::device, d_datasetSequence, d_datasetSequence + DATASET_COUNT);
  thrust::copy(thrust::device, d_cluster, d_cluster + DATASET_COUNT, d_tempCluster);
  thrust::sort_by_key(thrust::device, d_tempCluster, d_tempCluster + DATASET_COUNT, d_datasetSequence);

  thrust::pair<int *, int *> dataPositioned;
  dataPositioned = thrust::equal_range(thrust::device, d_tempCluster, d_tempCluster + DATASET_COUNT, UNPROCESSED);
  int first = dataPositioned.first - d_tempCluster;
  int last = dataPositioned.second  - d_tempCluster;
  
  int* datasetSequence = (int*)malloc(sizeof(int) * DATASET_COUNT);
  gpuErrchk(hipMemcpy(datasetSequence, d_datasetSequence, sizeof(int) * DATASET_COUNT, hipMemcpyDeviceToHost));
  
  int blockCount = 0;
  for(int x = first; x < last; x++) {
    if(blockCount > THREAD_BLOCKS) break;
    localSeedList[blockCount * MAX_SEEDS] = datasetSequence[x];
    localSeedLength[blockCount] = 1;
    blockCount++;
  }

  hipFree(d_datasetSequence);
  hipFree(d_tempCluster);
  free(datasetSequence);

  //////////////////////////////////////////////////////////////////////////////////////////

  // Finally, transfer back the CPU memory to GPU and run DBSCAN process

  gpuErrchk(hipMemcpy(d_seedLength, localSeedLength,
                       sizeof(int) * THREAD_BLOCKS, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_seedList, localSeedList,
                       sizeof(int) * THREAD_BLOCKS * MAX_SEEDS,
                       hipMemcpyHostToDevice));

  // Free CPU memories
  free(localSeedList);
  free(localSeedLength);

  if(first == last) {
    return true;
  }

  return false;
}