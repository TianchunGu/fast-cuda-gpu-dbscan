#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include <vector>

#define DATASET_COUNT 1000
#define DIMENSION 2
#define PARTITION 100
#define POINTS_SEARCHED 100
#define THREAD_BLOCKS 64
#define THREAD_COUNT 128

#define EPSILON 1.5

using namespace std;

struct __align__(8) dataNode {
    int id;
    struct dataNode *child;
};

struct __align__(8) IndexStructure {
    int level;
    double range[2];
    struct IndexStructure *buckets[PARTITION];
    struct dataNode *dataRoot;
};

#define gpuErrchk(ans) \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort) exit(code);
  }
}
 /**
   **************************************************************************
   * CUDA Data structures
   **************************************************************************
   */


__device__ double d_dataset[DATASET_COUNT*DIMENSION];
__device__ int d_results[POINTS_SEARCHED];
__device__ int d_partition[DIMENSION];
__device__ double d_minPoints[DIMENSION];

double * sym_dataset;
int * sym_results;
int * sym_partition;
double * sym_minPoints;

__global__ void Indexingkernel(struct IndexStructure *indexRoot, struct IndexStructure **indexBuckets, struct dataNode **dataNodes, int * indexBucketsLength, int * dataNodesLength);

int ImportDataset(char const *fname, double *dataset);

__device__ void indexConstruction(struct IndexStructure *indexRoot, struct IndexStructure **indexBuckets, int * indexBucketsLength, struct dataNode **dataNodes, int * dataNodesLength);

__device__
void insertData(int id, struct IndexStructure *indexRoot, struct dataNode **dataNodes, int * dataNodesLength);

__device__
void searchPoints(double *data, struct IndexStructure *indexRoot);

/**
//////////////////////////////////////////////////////////////////////////
**************************************************************************
* Main Function
**************************************************************************
//////////////////////////////////////////////////////////////////////////
*/
int main(int argc, char **argv) {
    char inputFname[500];
    if (argc != 2) {
        fprintf(stderr, "Please provide the dataset file path in the arguments\n");
        exit(0);
    }

    // Get the dataset file name from argument
    strcpy(inputFname, argv[1]);
    printf("Using dataset file %s\n", inputFname);

    double *importedDataset =
        (double *)malloc(sizeof(double) * DATASET_COUNT * DIMENSION);

    // Import data from dataset
    int ret = ImportDataset(inputFname, importedDataset);
    if (ret == 1) {
        printf("\nError importing the dataset");
        return 0;
    }  
    /**
   **************************************************************************
   * CUDA Memory allocation
   **************************************************************************
   */

   // Check if the GPU is functioning correctly
   gpuErrchk(hipFree(0));

   gpuErrchk(hipGetSymbolAddress((void **)&sym_dataset, d_dataset));
   gpuErrchk(hipGetSymbolAddress((void **)&sym_partition, d_partition));
   gpuErrchk(hipGetSymbolAddress((void **)&sym_results, d_results));
   gpuErrchk(hipGetSymbolAddress((void **)&sym_minPoints, d_minPoints)); 

    struct IndexStructure *d_indexRoot;
    gpuErrchk(hipMalloc((void **)&d_indexRoot, sizeof(struct IndexStructure)));

   /**
   **************************************************************************
   * CPU Memory allocation
   **************************************************************************
   */
    double maxPoints[DIMENSION];
    double minPoints[DIMENSION];

    for (int j = 0; j < DIMENSION; j++) {
        maxPoints[j] = 0;
        minPoints[j] = 999999999;
    }

    for (int i = 0; i < DATASET_COUNT; i++) {
        for (int j = 0; j < DIMENSION; j++) {
            if (importedDataset[i * DIMENSION + j] > maxPoints[j]) {
                maxPoints[j] = importedDataset[i * DIMENSION + j];
            }
            if (importedDataset[i * DIMENSION + j] < minPoints[j]) {
                minPoints[j] = importedDataset[i * DIMENSION + j];
            }
        }
    }

    int *partition = (int *)malloc(sizeof(int) * DIMENSION);

    for (int i = 0; i < DIMENSION; i++) {
        partition[i] = 0;
        double curr = minPoints[i];
        while (curr < maxPoints[i]) {
            partition[i]++;
            curr += EPSILON;
        }
    }

    /**
   **************************************************************************
   * Copy data from cpu to gpu
   **************************************************************************
   */

   gpuErrchk(hipMemcpy(sym_dataset, importedDataset,
    sizeof(double) * DATASET_COUNT * DIMENSION,
    hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(sym_minPoints, minPoints,
        sizeof(double) * DIMENSION,
        hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(sym_partition, partition,
            sizeof(int) * DIMENSION,
            hipMemcpyHostToDevice));

    gpuErrchk(hipMemset(sym_results, -1, sizeof(int) * POINTS_SEARCHED));

    int indexedStructureSize = 1;
    for(int i = 0; i < DIMENSION; i++) {
        indexedStructureSize *= partition[i];
    }

    int * d_indexBucketsLength, * d_dataNodesLength;
    gpuErrchk(hipMalloc((void **)&d_indexBucketsLength,sizeof(int)));
    gpuErrchk(hipMalloc((void **)&d_dataNodesLength, sizeof(int)));

    gpuErrchk(hipMemset(d_indexBucketsLength, 0, sizeof(int)));
    gpuErrchk(hipMemset(d_dataNodesLength, 0, sizeof(int)));
    
    
    // Allocate memory for index buckets
    struct IndexStructure **d_indexBuckets, *d_currentIndexBucket;

    gpuErrchk(hipMalloc((void **)&d_indexBuckets,sizeof(struct IndexStructure*) * indexedStructureSize*2));

    for(int i = 0; i < indexedStructureSize*2; i++) {
        gpuErrchk(hipMalloc((void **)&d_currentIndexBucket, sizeof(struct IndexStructure)));
        gpuErrchk(hipMemcpy(&d_indexBuckets[i], &d_currentIndexBucket, sizeof(struct IndexStructure*), hipMemcpyHostToDevice));
        
    }

    // Allocate memory for data Nodes
    struct dataNode **d_dataNodes, *d_currentdataNode;

    gpuErrchk(hipMalloc((void **)&d_dataNodes,sizeof(struct dataNode*) * DATASET_COUNT*5));

    for(int i = 0; i < DATASET_COUNT*5; i++) {
        gpuErrchk(hipMalloc((void **)&d_currentdataNode,sizeof(struct dataNode*)));
        gpuErrchk(hipMemcpy(&d_dataNodes[i], &d_currentdataNode, sizeof(struct dataNode*), hipMemcpyHostToDevice));
    }

     /**
   **************************************************************************
   * kernel Function...
   **************************************************************************
   */

   
   cudaThreadSetLimit(hipLimitMallocHeapSize, 128*1024*1024);
   Indexingkernel<<<dim3(THREAD_BLOCKS, 1), dim3(THREAD_COUNT, 1)>>>( d_indexRoot, d_indexBuckets, d_dataNodes, d_indexBucketsLength, d_dataNodesLength);

   gpuErrchk(hipDeviceSynchronize());

   /**
   **************************************************************************
   * Free CUDA memory allocations
   **************************************************************************
   */

    hipFree(d_indexRoot);
    
    return 0;
}

__global__ void Indexingkernel(struct IndexStructure *indexRoot, struct IndexStructure **indexBuckets, struct dataNode **dataNodes, int * indexBucketsLength, int * dataNodesLength) {

    if (threadIdx.x == 0 && blockIdx.x == 0) {
        indexConstruction(indexRoot, indexBuckets, indexBucketsLength, dataNodes, dataNodesLength);
    }
    __syncthreads();

    
    for(int i = threadIdx.x; i < DATASET_COUNT; i = i + THREAD_COUNT) {
        insertData(i, indexRoot, dataNodes, dataNodesLength);
    }

    /*
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        double data[DIMENSION];
        data[0] = d_dataset[0];
        data[0] = d_dataset[1];
        searchPoints(data, indexRoot);
    }

    __syncthreads();
    */
    

}


__device__ void indexConstruction(struct IndexStructure *indexRoot, struct IndexStructure **indexBuckets, int * indexBucketsLength, struct dataNode **dataNodes, int * dataNodesLength) {

    int indexedStructureSize = 1;
    for(int i = 0; i < DIMENSION; i++) {
      indexedStructureSize *= d_partition[i];
    }

    struct IndexStructure ** indexedStructures = (struct IndexStructure**) malloc(sizeof(struct IndexStructure*) * indexedStructureSize);

    int indexedStructureSizeCount = 0;
    indexedStructures[indexedStructureSizeCount++] = indexRoot;

    for (int j = 0; j < DIMENSION; j++) {

        struct IndexStructure ** childIndexedStructures = (struct IndexStructure**) malloc(sizeof(struct IndexStructure*) * indexedStructureSize);
        int childIndexedStructureSizeCount = 0;

        while (indexedStructureSizeCount > 0) {

            struct IndexStructure *currentIndex = indexedStructures[--indexedStructureSizeCount];
            currentIndex->level = j;

            double rightPoint = d_minPoints[j] + d_partition[j] * EPSILON;

            for (int k = d_partition[j] - 1; k >= 0; k--) {
                register int currentBucketCount = atomicAdd(indexBucketsLength, 1);

                struct IndexStructure *currentBucket = indexBuckets[currentBucketCount];

                currentBucket->range[1] = rightPoint;
                rightPoint = rightPoint - EPSILON;
                currentBucket->range[0] = rightPoint;

                if(j == DIMENSION -1) {
                    register int currentDataNodeCount = atomicAdd(dataNodesLength, 1);
                    currentBucket->dataRoot = dataNodes[currentDataNodeCount];
                    currentBucket->dataRoot->id = -1;
                }

                currentIndex->buckets[k] = currentBucket;
                if (j < DIMENSION - 1) {
                    childIndexedStructures[childIndexedStructureSizeCount++] = currentIndex->buckets[k];
                }
            }
        }

        while (childIndexedStructureSizeCount > 0) {
          indexedStructures[indexedStructureSizeCount++] = childIndexedStructures[--childIndexedStructureSizeCount];

          free(childIndexedStructures[childIndexedStructureSizeCount]);
        }

        free(childIndexedStructures);
    }

    for(int i = 0; i < indexedStructureSize; i++) {
        free(indexedStructures[i]);
    }
    free(indexedStructures);
}

__device__
void insertData(int id, struct IndexStructure *indexRoot, struct dataNode **dataNodes, int * dataNodesLength) {

    register float data[DIMENSION];
    for (int j = 0; j < DIMENSION; j++) {
        data[j] = d_dataset[id * DIMENSION + j];
    }

    struct IndexStructure *currentIndex = (struct IndexStructure *)malloc(sizeof(struct IndexStructure));

    struct dataNode *selectedDataNode = (struct dataNode *)malloc(sizeof(struct dataNode));

    currentIndex = indexRoot;
    bool found = false;

    while (!found) {
        int dimension = currentIndex->level;
        for (int k = 0; k < d_partition[dimension]; k++) {
            struct IndexStructure *currentBucket = (struct IndexStructure *)malloc(sizeof(struct IndexStructure));
            currentBucket = currentIndex->buckets[k];

            float comparingData = (float)data[dimension];
            float leftRange = (float)currentBucket->range[0];
            float rightRange = (float)currentBucket->range[1];

            if (comparingData >= leftRange && comparingData <= rightRange) {
                if (dimension == DIMENSION - 1) {
                    selectedDataNode = currentBucket->dataRoot;
                    found = true;
                    break;
                }
                currentIndex = currentBucket;
                break;
            }
        }
    }

    if (selectedDataNode->id == -1) {
        selectedDataNode->id = id;
        register int currentDataNodeCount = atomicAdd(dataNodesLength, 1);
        
        selectedDataNode->child =  dataNodes[currentDataNodeCount];
        selectedDataNode->child->id = -1;
    } else {
        selectedDataNode = selectedDataNode->child;
        while (selectedDataNode->id != -1) {
            selectedDataNode = selectedDataNode->child;
        }
        selectedDataNode->id = id;
        register int currentDataNodeCount = atomicAdd(dataNodesLength, 1);
        
        selectedDataNode->child =  dataNodes[currentDataNodeCount];
        selectedDataNode->child->id = -1;
        printf("%d\n", currentDataNodeCount);
    }
}


__device__
void searchPoints(double *data, struct IndexStructure *indexRoot) {
    struct IndexStructure *currentIndex = (struct IndexStructure *)malloc(sizeof(struct IndexStructure));

    struct dataNode *selectedDataNode = (struct dataNode *)malloc(sizeof(struct dataNode));

    // Size of data Node and index
    int indexedStructureSize = 1;
    for(int i = 0; i < DIMENSION; i++) {
      indexedStructureSize *= 3;
    }

    // Current Index
    struct IndexStructure ** currentIndexes = (struct IndexStructure**) malloc(sizeof(struct IndexStructure*) * indexedStructureSize);

    int currentIndexSize = 0;
    currentIndexes[currentIndexSize++] = indexRoot;

    // Selected data Node 
    struct dataNode ** selectedDataNodes = (struct dataNode **) malloc(sizeof(struct dataNode *) * indexedStructureSize);

    int selectedDataNodeSize = 0;

    while (currentIndexSize > 0) {

        currentIndex = currentIndexes[--currentIndexSize];

        int dimension = currentIndex->level;

        for (int k = 0; k < d_partition[dimension]; k++) {

            struct IndexStructure *currentBucket = (struct IndexStructure *)malloc(sizeof(struct IndexStructure));

            currentBucket = currentIndex->buckets[k];

            float comparingData = (float)data[dimension];
            float leftRange = (float)currentBucket->range[0];
            float rightRange = (float)currentBucket->range[1];

            if (comparingData >= leftRange && comparingData <= rightRange) {
                if (dimension == DIMENSION - 1) {
                    selectedDataNodes[selectedDataNodeSize++] = currentBucket->dataRoot;
                    if (k > 0) {
                        selectedDataNodes[selectedDataNodeSize++] = currentIndex->buckets[k - 1]->dataRoot;
                    }
                    if (k < d_partition[dimension] - 1) {
                        selectedDataNodes[selectedDataNodeSize++] = currentIndex->buckets[k + 1]->dataRoot;
                    }
                    break;
                }
                currentIndexes[currentIndexSize++] = currentBucket;
                if (k > 0) {
                    currentIndexes[currentIndexSize++] = currentIndex->buckets[k - 1];
                }
                if (k < d_partition[dimension] - 1) {
                    currentIndexes[currentIndexSize++] = currentIndex->buckets[k + 1];
                }
                break;
            }
        }
    }

    int resultsCount = 0;
    for (int x = 0; x < selectedDataNodeSize; x++) {
        selectedDataNode = selectedDataNodes[x];
        while (selectedDataNode->id != -1) {
            d_results[resultsCount++] = selectedDataNode->id;
            selectedDataNode = selectedDataNode->child;
        }
    }
}


int ImportDataset(char const *fname, double *dataset) {
    FILE *fp = fopen(fname, "r");
    if (!fp) {
        printf("Unable to open file\n");
        return (1);
    }

    char buf[4096];
    unsigned long int cnt = 0;
    while (fgets(buf, 4096, fp) && cnt < DATASET_COUNT * DIMENSION) {
        char *field = strtok(buf, ",");
        long double tmp;
        sscanf(field, "%Lf", &tmp);
        dataset[cnt] = tmp;
        cnt++;

        while (field) {
            field = strtok(NULL, ",");

            if (field != NULL) {
                long double tmp;
                sscanf(field, "%Lf", &tmp);
                dataset[cnt] = tmp;
                cnt++;
            }
        }
    }
    fclose(fp);
    return 0;
}