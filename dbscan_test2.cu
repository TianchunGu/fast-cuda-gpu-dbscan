#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <time.h>

#include <algorithm>
#include <ctime>
#include <fstream>
#include <map>
#include <math.h>
#include <set>
#include <vector>

using namespace std;

// Number of data in dataset to use
// #define DATASET_COUNT 1864620
#define DATASET_COUNT 10000

// Dimension of the dataset
#define DIMENSION 2

// Maximum size of seed list
#define MAX_SEEDS 256

// Extra collission size to detect final clusters collision
#define EXTRA_COLLISION_SIZE 1024

// Number of blocks
#define THREAD_BLOCKS 128

// Number of threads per block
#define THREAD_COUNT 256

// Status of points that are not clusterized
#define UNPROCESSED -1

// Status for noise point
#define NOISE -2

// Minimum number of points in DBSCAN
#define MINPTS 4

#define TREE_LEVELS (DIMENSION + 1)

// Epslion value in DBSCAN
#define EPS 1.5

#define RANGE 2

#define POINTS_SEARCHED 9

#define PARTITION_SIZE 50

/**
**************************************************************************
//////////////////////////////////////////////////////////////////////////
* GPU ERROR function checks for potential erros in cuda function execution
//////////////////////////////////////////////////////////////////////////
**************************************************************************
*/

#define gpuErrchk(ans) \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort) exit(code);
  }
}

/**
**************************************************************************
//////////////////////////////////////////////////////////////////////////
* INDEXING datastructure and functions
//////////////////////////////////////////////////////////////////////////
**************************************************************************
*/

struct __align__(8) IndexStructure {
  int id;
  int dimension;
  int dataBegin;
  int dataEnd;
  int childBuckets[PARTITION_SIZE];
};

__global__ void INDEXING_STRUCTURE(double *dataset, int *indexTreeMetaData,
                                   double *minPoints, double *binWidth,
                                   int *results,
                                   struct IndexStructure **indexBuckets,
                                   int *dataKey, int *dataValue, double * upperBounds);

__global__ void INDEXING_ADJUSTMENT(int *indexTreeMetaData,
                                    struct IndexStructure **indexBuckets,
                                    int *dataKey);

__device__ void indexConstruction(int dimension, int *indexTreeMetaData,
                                  double *minPoints, double *binWidth,
                                  struct IndexStructure **indexBuckets, double * upperBounds);

__device__ void insertData(int id, double *dataset,
                           struct IndexStructure **indexBuckets, int *dataKey,
                           int *dataValue, double * upperBounds,  double *binWidth);

__device__ void searchPoints(double *data, int chainID, double *dataset,
                             int *results, struct IndexStructure **indexBuckets,

                             int *indexesStack, int *dataValue);

void findIndexingBins(double *d_dataset, int *d_results, int *d_seedList, int *dataValue, int *dataKey, int *indexTreeMetaData, double *d_upperBounds);

int compare(const void *a, const void *b) { return (*(int *)a - *(int *)b); }

int compareDouble(const void *a, const void *b) {
  return (*(double *)a - *(double *)b);
}

/**
**************************************************************************
//////////////////////////////////////////////////////////////////////////
* Declare CPU and GPU Functions
//////////////////////////////////////////////////////////////////////////
**************************************************************************
*/
int ImportDataset(char const *fname, double *dataset);

bool MonitorSeedPoints(vector<int> &unprocessedPoints, map<int, set<int>> &collisionUnion, int *runningCluster,
                       int *d_cluster, int *d_seedList, int *d_seedLength,
                       int *d_collisionMatrix, int *d_extraCollision,
                       int *d_results);

void GetDbscanResult(int *d_cluster, int *runningCluster, int *clusterCount, int *noiseCount);

__global__ void DBSCAN(double *dataset, int *cluster, int *seedList,
                       int *seedLength, int *collisionMatrix,
                       int *extraCollision, int *results,
                       struct IndexStructure **indexBuckets,

                       int *indexesStack, int *dataValue);

__device__ void MarkAsCandidate(int neighborID, int chainID, int *cluster,
                                int *seedList, int *seedLength,
                                int *collisionMatrix, int *extraCollision);
/**
**************************************************************************
//////////////////////////////////////////////////////////////////////////
* Main CPU function
//////////////////////////////////////////////////////////////////////////
**************************************************************************
*/
int main(int argc, char **argv) {
  char inputFname[500];
  if (argc != 2) {
    fprintf(stderr, "Please provide the dataset file path in the arguments\n");
    exit(0);
  }

  // Get the dataset file name from argument
  strcpy(inputFname, argv[1]);
  printf("Using dataset file %s\n", inputFname);

  double *importedDataset =
      (double *)malloc(sizeof(double) * DATASET_COUNT * DIMENSION);

  // Import data from dataset
  int ret = ImportDataset(inputFname, importedDataset);
  if (ret == 1) {
    printf("\nError importing the dataset");
    return 0;
  }

  // Check if the data parsed is correct
  for (int i = 0; i < DIMENSION; i++) {
    printf("Sample Data %f\n", importedDataset[i]);
  }

  // Get the total count of dataset
  vector<int> unprocessedPoints;
  for (int x = 0; x < DATASET_COUNT; x++) {
    unprocessedPoints.push_back(x);
  }

  printf("Preprocessed %lu data in dataset\n", unprocessedPoints.size());

  // Reset the GPU device for potential memory issues
  gpuErrchk(hipDeviceReset());
  gpuErrchk(hipFree(0));

  // Start the time
  clock_t totalTimeStart, totalTimeStop, indexingStart, indexingStop;
  float totalTime = 0.0;
  float indexingTime = 0.0;
  totalTimeStart = clock();

  /**
 **************************************************************************
 * CUDA Memory allocation
 **************************************************************************
 */
  double *d_dataset;
  int *d_cluster;
  int *d_seedList;
  int *d_seedLength;
  int *d_collisionMatrix;
  int *d_extraCollision;

  gpuErrchk(hipMalloc((void **)&d_dataset,
                       sizeof(double) * DATASET_COUNT * DIMENSION));

  gpuErrchk(hipMalloc((void **)&d_cluster, sizeof(int) * DATASET_COUNT));

  gpuErrchk(hipMalloc((void **)&d_seedList,
                       sizeof(int) * THREAD_BLOCKS * MAX_SEEDS));

  gpuErrchk(hipMalloc((void **)&d_seedLength, sizeof(int) * THREAD_BLOCKS));

  gpuErrchk(hipMalloc((void **)&d_collisionMatrix,
                       sizeof(int) * THREAD_BLOCKS * THREAD_BLOCKS));

  gpuErrchk(hipMalloc((void **)&d_extraCollision,
                       sizeof(int) * THREAD_BLOCKS * EXTRA_COLLISION_SIZE));

  /**
 **************************************************************************
 * Indexing Memory allocation
 **************************************************************************
 */

  indexingStart = clock();

  int *d_indexTreeMetaData;
  int *d_results;
  double *d_minPoints;
  double *d_binWidth;

  gpuErrchk(hipMalloc((void **)&d_indexTreeMetaData,
                       sizeof(int) * TREE_LEVELS * RANGE));

  gpuErrchk(hipMalloc((void **)&d_results,
                       sizeof(int) * THREAD_BLOCKS * POINTS_SEARCHED));

  gpuErrchk(hipMalloc((void **)&d_minPoints, sizeof(double) * DIMENSION));

  gpuErrchk(hipMalloc((void **)&d_binWidth, sizeof(double) * DIMENSION));

  gpuErrchk(
      hipMemset(d_results, -1, sizeof(int) * THREAD_BLOCKS * POINTS_SEARCHED));

  /**
 **************************************************************************
 * Assignment with default values
 **************************************************************************
 */
  gpuErrchk(hipMemcpy(d_dataset, importedDataset,
                       sizeof(double) * DATASET_COUNT * DIMENSION,
                       hipMemcpyHostToDevice));

  gpuErrchk(hipMemset(d_cluster, UNPROCESSED, sizeof(int) * DATASET_COUNT));

  gpuErrchk(
      hipMemset(d_seedList, -1, sizeof(int) * THREAD_BLOCKS * MAX_SEEDS));

  gpuErrchk(hipMemset(d_seedLength, 0, sizeof(int) * THREAD_BLOCKS));

  gpuErrchk(hipMemset(d_collisionMatrix, -1,
                       sizeof(int) * THREAD_BLOCKS * THREAD_BLOCKS));

  gpuErrchk(hipMemset(d_extraCollision, -1,
                       sizeof(int) * THREAD_BLOCKS * EXTRA_COLLISION_SIZE));

  /**
**************************************************************************
* Initialize index structure
**************************************************************************
*/
  double maxPoints[DIMENSION];
  double minPoints[DIMENSION];

  for (int j = 0; j < DIMENSION; j++) {
    maxPoints[j] = 0;
    minPoints[j] = 999999999;
  }

  for (int i = 0; i < DATASET_COUNT; i++) {
    for (int j = 0; j < DIMENSION; j++) {
      if (importedDataset[i * DIMENSION + j] > maxPoints[j]) {
        maxPoints[j] = importedDataset[i * DIMENSION + j];
      }
      if (importedDataset[i * DIMENSION + j] < minPoints[j]) {
        minPoints[j] = importedDataset[i * DIMENSION + j];
      }
    }
  }

  for (int i = 0; i < DIMENSION; i++) {
    printf("Level %d Max: %f\n", i, maxPoints[i]);
    printf("Level %d Min: %f\n", i, minPoints[i]);
  }

  double binWidth[DIMENSION];
  double minBinSize = 99999999;
  for (int x = 0; x < DIMENSION; x++) {
    binWidth[x] = (double)(maxPoints[x] - minPoints[x]) / PARTITION_SIZE;
    if (minBinSize >= binWidth[x]) {
      minBinSize = binWidth[x];
    }
  }
  for (int x = 0; x < DIMENSION; x++) {
    printf("#%d Bin Width: %lf\n", x, binWidth[x]);
  }
  printf("Min Bin Size: %lf\n", minBinSize);

  if (minBinSize < EPS) {
    printf("Bin width (%f) is less than EPS(%f).", minBinSize, EPS);
    exit(0);
  }

  // Level Partition
  int treeLevelPartition[TREE_LEVELS] = {1};

  for (int i = 0; i < DIMENSION; i++) {
    treeLevelPartition[i + 1] = PARTITION_SIZE;
  }

  int childItems[TREE_LEVELS];
  int startEndIndexes[TREE_LEVELS * RANGE];

  int mulx = 1;
  for (int k = 0; k < TREE_LEVELS; k++) {
    mulx *= treeLevelPartition[k];
    childItems[k] = mulx;
  }

  for (int i = 0; i < TREE_LEVELS; i++) {
    if (i == 0) {
      startEndIndexes[i * RANGE + 0] = 0;
      startEndIndexes[i * RANGE + 1] = 1;
      continue;
    }
    startEndIndexes[i * RANGE + 0] = startEndIndexes[((i - 1) * RANGE) + 1];
    startEndIndexes[i * RANGE + 1] = startEndIndexes[i * RANGE + 0];
    for (int k = 0; k < childItems[i - 1]; k++) {
      startEndIndexes[i * RANGE + 1] += treeLevelPartition[i];
    }
  }

  for (int i = 0; i < TREE_LEVELS; i++) {
    printf("#%d ", i);
    printf("Partition: %d ", treeLevelPartition[i]);
    printf("Range: %d %d\n", startEndIndexes[i * RANGE + 0],
           startEndIndexes[i * RANGE + 1]);
  }

  gpuErrchk(hipMemcpy(d_minPoints, minPoints, sizeof(double) * DIMENSION,
                       hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_binWidth, binWidth, sizeof(double) * DIMENSION,
                       hipMemcpyHostToDevice));

  gpuErrchk(hipMemcpy(d_indexTreeMetaData, startEndIndexes,
                       sizeof(int) * TREE_LEVELS * RANGE,
                       hipMemcpyHostToDevice));

  int indexedStructureSize = startEndIndexes[DIMENSION * RANGE + 1];

  printf("Index Structure Size: %lf GB.\n",
         (sizeof(struct IndexStructure) * indexedStructureSize) /
             (1024 * 1024 * 1024.0));

  // Allocate memory for index buckets
  struct IndexStructure **d_indexBuckets, *d_currentIndexBucket;

  gpuErrchk(hipMalloc((void **)&d_indexBuckets,
                       sizeof(struct IndexStructure *) * indexedStructureSize));

  for (int i = 0; i < indexedStructureSize; i++) {
    gpuErrchk(hipMalloc((void **)&d_currentIndexBucket,
                         sizeof(struct IndexStructure)));
    gpuErrchk(hipMemcpy(&d_indexBuckets[i], &d_currentIndexBucket,
                         sizeof(struct IndexStructure *),
                         hipMemcpyHostToDevice));
  }

  // Allocate memory for current indexes stack
  int indexBucketSize = 1;
  for (int i = 0; i < DIMENSION; i++) {
    indexBucketSize *= 3;
  }

  indexBucketSize = indexBucketSize * THREAD_BLOCKS;

  int *d_indexesStack;

  gpuErrchk(
      hipMalloc((void **)&d_indexesStack, sizeof(int) * indexBucketSize));

  hipFree(d_currentIndexBucket);

  /**
 **************************************************************************
 * Data key-value pair
 **************************************************************************
 */
  int *d_dataKey;
  int *d_dataValue;
  double *d_upperBounds;

  gpuErrchk(hipMalloc((void **)&d_dataKey, sizeof(int) * DATASET_COUNT));
  gpuErrchk(hipMalloc((void **)&d_dataValue, sizeof(int) * DATASET_COUNT));
  gpuErrchk(hipMalloc((void **)&d_upperBounds, sizeof(double) * indexedStructureSize));

  /**
 **************************************************************************
 * Start Indexing first
 **************************************************************************
 */
  gpuErrchk(hipDeviceSynchronize());

  INDEXING_STRUCTURE<<<dim3(THREAD_BLOCKS, 1), dim3(THREAD_COUNT, 1)>>>(
      d_dataset, d_indexTreeMetaData, d_minPoints, d_binWidth, d_results,
      d_indexBuckets, d_dataKey, d_dataValue, d_upperBounds);
  gpuErrchk(hipDeviceSynchronize());

  hipFree(d_indexTreeMetaData);
  hipFree(d_minPoints);

  /**
 **************************************************************************
 * Sorting and adjusting Data key-value pair
 **************************************************************************
 */

  thrust::sort_by_key(thrust::device, d_dataKey, d_dataKey + DATASET_COUNT,
                      d_dataValue);

  gpuErrchk(hipDeviceSynchronize());

  INDEXING_ADJUSTMENT<<<dim3(THREAD_BLOCKS, 1), dim3(THREAD_COUNT, 1)>>>(
      d_indexTreeMetaData, d_indexBuckets, d_dataKey);

  gpuErrchk(hipDeviceSynchronize());

  indexingStop = clock();

  printf("Index structure created.\n");

  /**
 **************************************************************************
 * Start the DBSCAN algorithm
 **************************************************************************
 */

  // Keep track of number of cluster formed without global merge
  int runningCluster = THREAD_BLOCKS;
  map<int, set<int>> collisionUnion;

  // Global cluster count
  int clusterCount = 0;

  // Keeps track of number of noises
  int noiseCount = 0;

  // Handler to conmtrol the while loop
  bool exit = false;

  while (!exit) {
    // Monitor the seed list and return the comptetion status of points
    int completed = MonitorSeedPoints(
        unprocessedPoints, collisionUnion, &runningCluster, d_cluster, d_seedList, d_seedLength,
        d_collisionMatrix, d_extraCollision, d_results);

    // printf("Running cluster %d, unprocessed points: %lu\n", runningCluster,
    //     unprocessedPoints.size());

    // If all points are processed, exit
    if (completed) {
      exit = true;
    }

    if (exit) break;

    // Kernel function to expand the seed list
    gpuErrchk(hipDeviceSynchronize());
    DBSCAN<<<dim3(THREAD_BLOCKS, 1), dim3(THREAD_COUNT, 1)>>>(
        d_dataset, d_cluster, d_seedList, d_seedLength, d_collisionMatrix,
        d_extraCollision, d_results, d_indexBuckets, d_indexesStack,
        d_dataValue);
    gpuErrchk(hipDeviceSynchronize());
  }

  /**
 **************************************************************************
 * End DBSCAN and show the results
 **************************************************************************
 */
  
 printf("DBSCAN completed. Finalizing clusters...\n");

  // Get the DBSCAN result
  GetDbscanResult(d_cluster, &runningCluster, &clusterCount, &noiseCount);

  totalTimeStop = clock();
  totalTime = (float)(totalTimeStop - totalTimeStart) / CLOCKS_PER_SEC;
  indexingTime = (float)(indexingStop - indexingStart) / CLOCKS_PER_SEC;

  printf("==============================================\n");
  printf("Final cluster after merging: %d\n", clusterCount);
  printf("Number of noises: %d\n", noiseCount);
  printf("==============================================\n");
  printf("Indexing Time: %3.2f seconds\n", indexingTime);
  printf("Total Time: %3.2f seconds\n", totalTime);
  printf("==============================================\n");

  /**
 **************************************************************************
 * Free CUDA memory allocations
 **************************************************************************
 */

  hipFree(d_dataset);
  hipFree(d_cluster);
  hipFree(d_seedList);
  hipFree(d_seedLength);
  hipFree(d_collisionMatrix);
  hipFree(d_extraCollision);

  hipFree(d_results);
  hipFree(d_indexBuckets);
  hipFree(d_indexesStack);

  hipFree(d_dataKey);
  hipFree(d_dataValue);
}

/**
**************************************************************************
//////////////////////////////////////////////////////////////////////////
* Monitor Seed Points performs the following operations.
* 1) Check if the seed list is empty. If it is empty check the refill seed list
* else, return false to process next seed point by DBSCAN.
* 2) If seed list is empty, It will check refill seed list and fill the points
* from refill seed list to seed list
* 3) If seed list and refill seed list both are empty, then check for the
* collision matrix and form a cluster by merging chains.
* 4) After clusters are merged, new points are assigned to seed list
* 5) Lastly, It checks if all the points are processed. If so it will return
* true and DBSCAN algorithm will exit.
//////////////////////////////////////////////////////////////////////////
**************************************************************************
*/

void findIndexingBins(double *d_dataset, int *d_results, int *d_seedList, int *dataValue, int *dataKey, int *indexTreeMetaData, double *d_upperBounds) {

  int *localSeedList;
  localSeedList = (int *)malloc(sizeof(int) * THREAD_BLOCKS * MAX_SEEDS);
  gpuErrchk(hipMemcpy(localSeedList, d_seedList,
                       sizeof(int) * THREAD_BLOCKS * MAX_SEEDS,
                       hipMemcpyDeviceToHost));

  int *localResults;
  localResults = (int *)malloc(sizeof(int) * THREAD_BLOCKS * POINTS_SEARCHED);
  gpuErrchk(hipMemcpy(localResults, d_results,
                      sizeof(int) * THREAD_BLOCKS * POINTS_SEARCHED,
                      hipMemcpyDeviceToHost));
  int* localResultsLength;
  localResultsLength = (int*)malloc(sizeof(int) * THREAD_BLOCKS);


  for(int z = 0; z < THREAD_BLOCKS; z++) {
    stack <int> s;
    s.push(0);
    localResultsLength[z] = 0;
    
    for(int x = 0; x < DIMENSION; x++) {
      while(!s.empty()) {
        int level = x + 1;
        // d_upperBounds
        // indexTreeMetaData
        int currentIndex = s.top();
        s.pop();
        double data = d_dataset[localSeedList[z*THREAD_BLOCKS] * DIMENSION + x];
        int position = thrust::upper_bound(thrust::device, d_upperBounds + currentIndex, d_upperBounds + currentIndex + PARTITION_SIZE, data) - d_upperBounds;
        s.push(position);
        s.push(position - 1);
        s.push(position + 1);

        if(level == DIMENSION ) {
          localResults[z*POINTS_SEARCHED + localResultsLength[z]++] = s.top();
          s.pop();
          localResults[z*POINTS_SEARCHED + localResultsLength[z]++] = s.top();
          s.pop();
          localResults[z*POINTS_SEARCHED + localResultsLength[z]++] = s.top();
          s.pop();
        }
      }
    }
  }

  for(int x = 0; x < THREAD_BLOCKS; x++) {
    for(int y = 0; y < POINTS_SEARCHED; y++) {
      cout << localResults[x + POINTS_SEARCHED + y] << " " ;
    }
    cout << endl;
  }
}

bool MonitorSeedPoints(vector<int> &unprocessedPoints, map<int, set<int>> &collisionUnion, int *runningCluster,
                       int *d_cluster, int *d_seedList, int *d_seedLength,
                       int *d_collisionMatrix, int *d_extraCollision,
                       int *d_results) {
  /**
 **************************************************************************
 * Copy GPU variables content to CPU variables for seed list management
 **************************************************************************
 */
  int *localSeedLength;
  localSeedLength = (int *)malloc(sizeof(int) * THREAD_BLOCKS);
  gpuErrchk(hipMemcpy(localSeedLength, d_seedLength,
                       sizeof(int) * THREAD_BLOCKS, hipMemcpyDeviceToHost));

  int *localSeedList;
  localSeedList = (int *)malloc(sizeof(int) * THREAD_BLOCKS * MAX_SEEDS);
  gpuErrchk(hipMemcpy(localSeedList, d_seedList,
                       sizeof(int) * THREAD_BLOCKS * MAX_SEEDS,
                       hipMemcpyDeviceToHost));

  /**
 **************************************************************************
 * Copy GPU variables to CPU variables for collision detection
 **************************************************************************
 */

  int *localCluster;
  localCluster = (int *)malloc(sizeof(int) * DATASET_COUNT);
  gpuErrchk(hipMemcpy(localCluster, d_cluster, sizeof(int) * DATASET_COUNT,
                       hipMemcpyDeviceToHost));

  int *localCollisionMatrix;
  localCollisionMatrix =
      (int *)malloc(sizeof(int) * THREAD_BLOCKS * THREAD_BLOCKS);
  gpuErrchk(hipMemcpy(localCollisionMatrix, d_collisionMatrix,
                       sizeof(int) * THREAD_BLOCKS * THREAD_BLOCKS,
                       hipMemcpyDeviceToHost));

  int *localExtraCollision;
  localExtraCollision =
      (int *)malloc(sizeof(int) * THREAD_BLOCKS * EXTRA_COLLISION_SIZE);
  gpuErrchk(hipMemcpy(localExtraCollision, d_extraCollision,
                       sizeof(int) * THREAD_BLOCKS * EXTRA_COLLISION_SIZE,
                       hipMemcpyDeviceToHost));

  /**
 **************************************************************************
 * If seedlist is empty and refill is also empty Then check the `
 * between chains and finalize the clusters
 **************************************************************************
 */

  map<int, int> clusterMap;
  set<int> blockSet;
 for (int i = 0; i < THREAD_BLOCKS; i++) {
    blockSet.insert(i);
  }

  set<int>::iterator it;

  while (blockSet.empty() == 0) {
    it = blockSet.begin();
    int curBlock = *it;
    set<int> expansionQueue;
    set<int> finalQueue;

    expansionQueue.insert(curBlock);
    finalQueue.insert(curBlock);

   while (expansionQueue.empty() == 0) {
      it = expansionQueue.begin();
      int expandBlock = *it;
      expansionQueue.erase(it);
      blockSet.erase(expandBlock);
      for (int x = 0; x < THREAD_BLOCKS; x++) {
        if (x == expandBlock) continue;
        if (localCollisionMatrix[expandBlock * THREAD_BLOCKS + x] == 1 &&
            blockSet.find(x) != blockSet.end()) {
          expansionQueue.insert(x);
          finalQueue.insert(x);
        }
      }
    }

    for (it = finalQueue.begin(); it != finalQueue.end(); ++it) {
      clusterMap[*it] = curBlock;
    }
  }


  map<int, int> clusterCountMap;
  for(int x = 0; x < THREAD_BLOCKS; x++) {
    if(clusterCountMap[clusterMap[x]] != 0) continue;
    clusterCountMap[clusterMap[x]] = (*runningCluster);
    (*runningCluster)++;
  }
  
  for (int i = 0; i < DATASET_COUNT; i++) {
    if (localCluster[i] >= 0 && localCluster[i] < THREAD_BLOCKS) {
      localCluster[i] = clusterCountMap[clusterMap[localCluster[i]]];
    }
  }

  gpuErrchk(hipMemcpy(d_cluster, localCluster, sizeof(int) * DATASET_COUNT,
                       hipMemcpyHostToDevice));

  for(int x = 0; x < THREAD_BLOCKS; x++) {
    if(localExtraCollision[x * EXTRA_COLLISION_SIZE] == -1) continue;
    int minCluster = localExtraCollision[x * EXTRA_COLLISION_SIZE];
    thrust::replace(thrust::device, d_cluster, d_cluster + DATASET_COUNT, clusterCountMap[clusterMap[x]], minCluster);
    for(int y = 0; y < EXTRA_COLLISION_SIZE; y++) {
      if(localExtraCollision[x * EXTRA_COLLISION_SIZE + y] == UNPROCESSED) break;
      int data = localExtraCollision[x * EXTRA_COLLISION_SIZE + y];
      thrust::replace(thrust::device, d_cluster, d_cluster + DATASET_COUNT, data, minCluster);
    }
  }

  /**
 **************************************************************************
 * After finilazing the cluster, check the remaining points and
 * insert one point to each of the seedlist
 **************************************************************************
 */

  int complete = 0;
  for (int i = 0; i < THREAD_BLOCKS; i++) {
    bool found = false;
    while (!unprocessedPoints.empty()) {
      int lastPoint = unprocessedPoints.back();
      unprocessedPoints.pop_back();

      if (localCluster[lastPoint] == UNPROCESSED) {
        localSeedLength[i] = 1;
        localSeedList[i * MAX_SEEDS] = lastPoint;
        found = true;
        break;
      }
    }

    if (!found) {
      complete++;
    }
  }

  /**
**************************************************************************
* FInally, transfer back the CPU memory to GPU and run DBSCAN process
**************************************************************************
*/

  gpuErrchk(hipMemcpy(d_seedLength, localSeedLength,
                       sizeof(int) * THREAD_BLOCKS, hipMemcpyHostToDevice));

  gpuErrchk(hipMemcpy(d_seedList, localSeedList,
                       sizeof(int) * THREAD_BLOCKS * MAX_SEEDS,
                       hipMemcpyHostToDevice));

  /**
 **************************************************************************
 * Free CPU memory allocations
 **************************************************************************
 */

  free(localCluster);
  free(localSeedList);
  free(localSeedLength);
  free(localCollisionMatrix);
  free(localExtraCollision);

  if (complete == THREAD_BLOCKS) {
    return true;
  }

  return false;
}

/**
**************************************************************************
//////////////////////////////////////////////////////////////////////////
* Get DBSCAN result
* Get the final cluster and print the overall result
//////////////////////////////////////////////////////////////////////////
**************************************************************************
*/
void GetDbscanResult(int *d_cluster, int *runningCluster,
                     int *clusterCount, int *noiseCount) {

int localClusterCount = 0;
for (int i = THREAD_BLOCKS; i < (*runningCluster); i++) {
  if(thrust::find(thrust::device, d_cluster, d_cluster + DATASET_COUNT, i) != d_cluster + DATASET_COUNT) {
    thrust::replace(thrust::device, d_cluster, d_cluster + DATASET_COUNT, i, ++localClusterCount);
  }
}
*clusterCount = localClusterCount;
*noiseCount = thrust::count(thrust::device, d_cluster, d_cluster + DATASET_COUNT, NOISE);
  
}

/**
**************************************************************************
//////////////////////////////////////////////////////////////////////////
* DBSCAN: Main kernel function of the algorithm
* It does the following functions.
* 1) Every block gets a point from seedlist to expand. If these points are
* processed already, it returns
* 2) It expands the points by finding neighbors points
* 3) Checks for the collision and mark the collision in collision matrix
//////////////////////////////////////////////////////////////////////////
**************************************************************************
*/
__global__ void DBSCAN(double *dataset, int *cluster, int *seedList,
                       int *seedLength, int *collisionMatrix,
                       int *extraCollision, int *results,
                       struct IndexStructure **indexBuckets,

                       int *indexesStack, int *dataValue) {
  /**
 **************************************************************************
 * Define shared variables
 **************************************************************************
 */

  // Point ID to expand by a block
  __shared__ int pointID;

  // Neighbors to store of neighbors points exceeds minpoints
  __shared__ int neighborBuffer[MINPTS];

  // It counts the total neighbors
  __shared__ int neighborCount;

  // ChainID is basically blockID
  __shared__ int chainID;

  // Store the point from pointID
  __shared__ double point[DIMENSION];

  // Length of the seedlist to check its size
  __shared__ int currentSeedLength;

  __shared__ int resultId;

  if (threadIdx.x == 0) {
    chainID = blockIdx.x;
    currentSeedLength = seedLength[chainID];
    pointID = seedList[chainID * MAX_SEEDS + currentSeedLength - 1];
  }
  __syncthreads();

  int threadId = blockDim.x * blockIdx.x + threadIdx.x;
  for(int x = threadId; x < THREAD_BLOCKS*THREAD_BLOCKS; x = x + THREAD_BLOCKS*THREAD_COUNT) {
    collisionMatrix[x] = UNPROCESSED;
  }
  for(int x = threadId; x < THREAD_BLOCKS*EXTRA_COLLISION_SIZE; x = x + THREAD_BLOCKS*THREAD_COUNT) {
    extraCollision[x] = UNPROCESSED;
  }

  __syncthreads();

  // Complete the seedlist to proceed.

  while (seedLength[chainID] != 0) {


      for(int x = threadId; x < THREAD_BLOCKS*POINTS_SEARCHED; x = x + THREAD_BLOCKS*THREAD_COUNT) {
        results[x] = UNPROCESSED;
      }
      __syncthreads();

    // Assign chainID, current seed length and pointID
    if (threadIdx.x == 0) {
      chainID = blockIdx.x;
      currentSeedLength = seedLength[chainID];
      pointID = seedList[chainID * MAX_SEEDS + currentSeedLength - 1];
    }
    __syncthreads();

    // Check if the point is already processed
    if (threadIdx.x == 0) {
      seedLength[chainID] = currentSeedLength - 1;
      neighborCount = 0;
      for (int x = 0; x < DIMENSION; x++) {
        point[x] = dataset[pointID * DIMENSION + x];
      }
    }
    __syncthreads();

    /**
**************************************************************************
* Find the neighbors of the pointID
* Mark point as candidate if points are more than min points
* Keep record of left over neighbors in neighborBuffer
**************************************************************************
*/

    for (int k = 0; k < POINTS_SEARCHED; k++) {


      if (threadIdx.x == 0) {
        resultId = results[chainID * POINTS_SEARCHED + k];
      }
      __syncthreads();


      if (resultId == -1) break;

      for (int i = threadIdx.x + indexBuckets[resultId]->dataBegin;
           i < indexBuckets[resultId]->dataEnd; i = i + THREAD_COUNT) {
        register double comparingPoint[DIMENSION];

        for (int x = 0; x < DIMENSION; x++) {
          comparingPoint[x] = dataset[dataValue[i] * DIMENSION + x];
        }

        register double distance = 0;
        for (int x = 0; x < DIMENSION; x++) {
          distance +=
              (point[x] - comparingPoint[x]) * (point[x] - comparingPoint[x]);
        }

        if (distance <= EPS * EPS) {
          register int currentNeighborCount = atomicAdd(&neighborCount, 1);
          if (currentNeighborCount >= MINPTS) {
            MarkAsCandidate(dataValue[i], chainID, cluster, seedList,
                            seedLength, collisionMatrix, extraCollision);
          } else {
            neighborBuffer[currentNeighborCount] = dataValue[i];
          }
        }
      }
      __syncthreads();
    }
    __syncthreads();

    /**
**************************************************************************
* Mark the left over neighbors in neighborBuffer as cluster member
* If neighbors are less than MINPTS, assign pointID with noise
**************************************************************************
*/

    if (neighborCount >= MINPTS) {
      cluster[pointID] = chainID;
      for (int i = threadIdx.x; i < MINPTS; i = i + THREAD_COUNT) {
        MarkAsCandidate(neighborBuffer[i], chainID, cluster, seedList,
                        seedLength, collisionMatrix, extraCollision);
      }
    } else {
      cluster[pointID] = NOISE;
    }

    __syncthreads();

    /**
**************************************************************************
* Check Thread length, If it exceeds MAX limit the length
* As seedlist wont have data beyond its max length
**************************************************************************
*/

    if (threadIdx.x == 0 && seedLength[chainID] >= MAX_SEEDS) {
      seedLength[chainID] = MAX_SEEDS - 1;
    }
    __syncthreads();
  }
}

/**
**************************************************************************
//////////////////////////////////////////////////////////////////////////
* Mark as candidate
* It does the following functions:
* 1) Mark the neighbor's cluster with chainID if its old state is unprocessed
* 2) If the oldstate is unprocessed, insert the neighnor point to seed list
* 3) if the seed list exceeds max value, insert into refill seed list
* 4) If the old state is less than THREAD BLOCK, record the collision in
* collision matrix
* 5) If the old state is greater than THREAD BLOCK, record the collision
* in extra collision
//////////////////////////////////////////////////////////////////////////
**************************************************************************
*/

__device__ void MarkAsCandidate(int neighborID, int chainID, int *cluster,
                                int *seedList, int *seedLength,
                                int *collisionMatrix, int *extraCollision) {
  /**
**************************************************************************
* Get the old cluster state of the neighbor
* If the state is unprocessed, assign it with chainID
**************************************************************************
*/
  register int oldState =
      atomicCAS(&(cluster[neighborID]), UNPROCESSED, chainID);

  /**
 **************************************************************************
 * For unprocessed old state of neighbors, add them to seedlist and
 * refill seedlist
 **************************************************************************
 */
  if (oldState == UNPROCESSED) {
    register int sl = atomicAdd(&(seedLength[chainID]), 1);
    if (sl < MAX_SEEDS) {
      seedList[chainID * MAX_SEEDS + sl] = neighborID;
    }
  }

  /**
 **************************************************************************
 * If the old state is greater than thread block, record the extra collisions
 **************************************************************************
 */

  else if (oldState >= THREAD_BLOCKS) {
    for (int i = 0; i < EXTRA_COLLISION_SIZE; i++) {
      register int changedState =
          atomicCAS(&(extraCollision[chainID * EXTRA_COLLISION_SIZE + i]),
                    UNPROCESSED, oldState);
      if (changedState == UNPROCESSED || changedState == oldState) {
        break;
      }
    }
  }

  /**
 **************************************************************************
 * If the old state of neighbor is not noise, not member of chain and cluster
 * is within THREADBLOCK, maek the collision between old and new state
 **************************************************************************
 */
  else if (oldState != NOISE && oldState != chainID &&
           oldState < THREAD_BLOCKS) {
    collisionMatrix[oldState * THREAD_BLOCKS + chainID] = 1;
    collisionMatrix[chainID * THREAD_BLOCKS + oldState] = 1;
  }

  /**
 **************************************************************************
 * If the old state is noise, assign it to chainID cluster
 **************************************************************************
 */
  else if (oldState == NOISE) {
    oldState = atomicCAS(&(cluster[neighborID]), NOISE, chainID);
  }
}

/**
**************************************************************************
//////////////////////////////////////////////////////////////////////////
* Helper functions for index construction and points search...
//////////////////////////////////////////////////////////////////////////
**************************************************************************
*/

__global__ void INDEXING_STRUCTURE(double *dataset, int *indexTreeMetaData,
                                   double *minPoints, double *binWidth,
                                   int *results,
                                   struct IndexStructure **indexBuckets,
                                   int *dataKey, int *dataValue, double * upperBounds) {
  if(blockIdx.x < DIMENSION) {
    indexConstruction(blockIdx.x, indexTreeMetaData, minPoints, binWidth, indexBuckets, upperBounds);
  }
  __syncthreads();

  int threadId = blockDim.x * blockIdx.x + threadIdx.x;
  for (int i = threadId; i < DATASET_COUNT;
       i = i + THREAD_COUNT * THREAD_BLOCKS) {
    insertData(i, dataset, indexBuckets, dataKey, dataValue, upperBounds, binWidth);
  }
  __syncthreads();
}

__global__ void INDEXING_ADJUSTMENT(int *indexTreeMetaData,
                                    struct IndexStructure **indexBuckets,
                                    int *dataKey) {
  __shared__ int indexingRange;
  if (threadIdx.x == 0) {
    indexingRange = indexTreeMetaData[DIMENSION * RANGE + 1] -
                    indexTreeMetaData[DIMENSION * RANGE];
  }
  __syncthreads();

  int threadId = blockDim.x * blockIdx.x + threadIdx.x;

  for (int i = threadId; i < indexingRange;
       i = i + THREAD_COUNT * THREAD_BLOCKS) {
    int idx = indexTreeMetaData[DIMENSION * RANGE] + i;

    thrust::pair<int *, int *> dataPositioned;

    dataPositioned = thrust::equal_range(thrust::device, dataKey,
                                         dataKey + DATASET_COUNT, idx);

    indexBuckets[idx]->dataBegin = dataPositioned.first - dataKey;
    indexBuckets[idx]->dataEnd = dataPositioned.second - dataKey;
  }
  __syncthreads();
}

__device__ void indexConstruction(int level, int *indexTreeMetaData,
                                  double *minPoints, double *binWidth,
                                  struct IndexStructure **indexBuckets, double * upperBounds) {
  
  
  for (int k = threadIdx.x + indexTreeMetaData[level * RANGE + 0];
       k < indexTreeMetaData[level * RANGE + 1]; k = k + THREAD_COUNT) {

    for (int i = 0; i < PARTITION_SIZE; i++) {
      
      int currentBucketIndex =
          indexTreeMetaData[level * RANGE + 1] + i +
          (k - indexTreeMetaData[level * RANGE + 0]) * PARTITION_SIZE;

      indexBuckets[k]->dimension = level;
      indexBuckets[currentBucketIndex]->dimension = level + 1;

      indexBuckets[currentBucketIndex]->id = currentBucketIndex;
      indexBuckets[k]->childBuckets[i] = currentBucketIndex;

      double rightPoint = minPoints[level] + i * binWidth[level] + binWidth[level];

      if (i == PARTITION_SIZE - 1) rightPoint = rightPoint + binWidth[level];

      upperBounds[currentBucketIndex] = rightPoint;

    }
  }
  __syncthreads();
}

__device__ void insertData(int id, double *dataset,
                           struct IndexStructure **indexBuckets, int *dataKey,
                           int *dataValue, double * upperBounds, double *binWidth) {
  double data[DIMENSION];
  for (int j = 0; j < DIMENSION; j++) {
    data[j] = dataset[id * DIMENSION + j];
  }

  int currentIndex = 0;
  bool found = false;

  while (!found) {
    if(indexBuckets[currentIndex]->dimension >= DIMENSION) break;
    for (int k = 0; k < PARTITION_SIZE; k++) {
      double comparingData = data[indexBuckets[currentIndex]->dimension];

      double leftRange;
      if(k == 0) {
        leftRange = upperBounds[indexBuckets[currentIndex]->childBuckets[k] - 1] - binWidth[indexBuckets[currentIndex]->dimension];
      } else {
        leftRange =
          upperBounds[indexBuckets[currentIndex]->childBuckets[k] - 1];
      }
      
      double rightRange = upperBounds[indexBuckets[currentIndex]->childBuckets[k]];

      printf("%f %f\n", leftRange, rightRange);
          

      if (comparingData >= leftRange && comparingData < rightRange) {
        if (indexBuckets[currentIndex]->dimension == DIMENSION - 1) {
          dataValue[id] = id;
          dataKey[id] = indexBuckets[currentIndex]->childBuckets[k];
          found = true;
          break;
        }
        currentIndex = indexBuckets[currentIndex]->childBuckets[k];
        break;
      }
    }
  }
}


/**
**************************************************************************
//////////////////////////////////////////////////////////////////////////
* Import Dataset
* It imports the data from the file and store in dataset variable
//////////////////////////////////////////////////////////////////////////
**************************************************************************
*/
int ImportDataset(char const *fname, double *dataset) {
  FILE *fp = fopen(fname, "r");
  if (!fp) {
    printf("Unable to open file\n");
    return (1);
  }

  char buf[4096];
  unsigned long int cnt = 0;
  while (fgets(buf, 4096, fp) && cnt < DATASET_COUNT * DIMENSION) {
    char *field = strtok(buf, ",");
    long double tmp;
    sscanf(field, "%Lf", &tmp);
    dataset[cnt] = tmp;
    cnt++;

    while (field) {
      field = strtok(NULL, ",");

      if (field != NULL) {
        long double tmp;
        sscanf(field, "%Lf", &tmp);
        dataset[cnt] = tmp;
        cnt++;
      }
    }
  }
  fclose(fp);
  return 0;
}
